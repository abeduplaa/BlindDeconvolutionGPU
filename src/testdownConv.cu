#include "downConvolution.cuh"
#include "testdownConv.cuh"
#include <iostream>


int downConvTest(){
    
    size_t nc = 3;
    size_t w = 3; 
    size_t h = 3;
    size_t m = 9; 
    size_t n = 9;
    size_t outSizeX = m-w+1;  
    size_t outSizeY = n-h+1;
    size_t inSize = w*h*nc;
    size_t outSize = outSizeX * outSizeY * nc;
    float* imgIn = new float[inSize];
    float* imgOut = new float[outSize];
    float* imgDownConv =  new float[outSize];
    float* kernel = new float[m*n];
    for(int i=0; i<inSize; ++i)
        imgIn[i] = 1.0f;
    for(int i=0; i<m*n; ++i)
        kernel[i] = 0.5f;

    //CPU commands
    //padImgCPU(imgOut, imgIn, w, h, nc, m, n);
    /*std::cout<<"padding done"<<std::endl;*/
    
    computeDownConvolutionCPU(imgDownConv, imgIn, kernel, w, h, nc, m, n);

    // //GPU commnds
    // float* d_imgIn = NULL;
    // float* d_imgOut = NULL;
    // float* d_kernel = NULL;
    // float* d_imgUpConv = NULL;
    // hipMalloc(&d_imgIn, inSize*sizeof(float));
    // hipMalloc(&d_imgOut, outSize*sizeof(float));
    // hipMalloc(&d_kernel, m*n*sizeof(float));
    // hipMalloc(&d_imgUpConv, outSize*sizeof(float));
    // hipMemcpy(d_imgIn, imgIn, inSize*sizeof(float), hipMemcpyHostToDevice);
    // hipMemcpy(d_kernel, kernel, m*n*sizeof(float), hipMemcpyHostToDevice);
    // padImgGlobalMemCuda(d_imgOut, d_imgIn, w, h, nc, m, n);
    // computeUpConvolutionGlobalMemCuda(d_imgUpConv, d_imgIn, d_kernel, w, h, nc, m, n);
    // hipMemcpy(imgOut, d_imgOut, outSize*sizeof(float), hipMemcpyDeviceToHost);
    // hipMemcpy(imgUpConv, d_imgUpConv, outSize*sizeof(float), hipMemcpyDeviceToHost);

    std::cout<<"Input Image"<<std::endl;
    for(int c=0; c<nc; ++c){
        std::cout<<"Channel no :  "<<c<<std::endl;
        for(int j=0; j<h; ++j){
            for(int i=0; i<w; ++i){
                std::cout<<imgIn[i + (j*w) + (c*w*h)]<<"   ";
            }
            std::cout<<std::endl;
        }
    }

    // std::cout<<"Output Image after padding"<<std::endl;
    // for(int c=0; c<nc; ++c){
    //     std::cout<<"Channel no :  "<<c<<std::endl;
    //     for(int j=0; j<outSizeY; ++j){
    //         for(int i=0; i<outSizeX; ++i){
    //             std::cout<<imgOut[i + (j*outSizeX) + (c*outSizeX*outSizeY)]<<"   ";
    //         }
    //         std::cout<<std::endl;
    //     }
    // }

    std::cout<<"After Down convolution"<<std::endl;
    for(int c=0; c<nc; ++c){
        std::cout<<"Channel no :  "<<c<<std::endl;
        for(int j=0; j<outSizeY; ++j){
            for(int i=0; i<outSizeX; ++i){
                std::cout<<imgDownConv[i + (j*outSizeX) + (c*outSizeX*outSizeY)]<<"   ";
            }
            std::cout<<std::endl;
        }
    }
    //Free Cuda Memory
    // hipFree(d_imgIn);
    // hipFree(d_imgOut);
    // hipFree(d_kernel);
    // hipFree(d_imgUpConv);

    //Free Host Memory
    delete[] imgIn;
    delete[] imgOut;
    delete[] imgDownConv;
    delete[] kernel;
    
}