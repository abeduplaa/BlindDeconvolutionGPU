#include <iostream>
// #include <algorithm>

#include "testdownConv.cuh"
#include "downConvolution.cuh"
#include "helper.cuh"
#include "epsilon.cuh"
#include "hipblas.h"

void downConvTest(){
    
    // Initialize variables
    size_t nc = 1;
    size_t w = 7; 
    size_t h = 7;
    size_t m = 5; 
    size_t n = 5;
    size_t inSizeX = n+1;
    size_t inSizeY = m+1;
    size_t outSizeX = n-w+1;  
    size_t outSizeY = m-h+1;
    size_t inSize = m*n*nc;
	size_t kernelSize = h*w;
    size_t outSize = outSizeX * outSizeY * nc;


     // Initialize Cublas
     hipblasHandle_t handle;
     hipblasCreate(&handle);

    float eps = 0;
    float epsCPU = 0;

    // allocate memory for arrays
    float* imgIn = new float[inSize];
    float* dummyGradU = new float[inSize];

    float* kernel = new float[kernelSize];

    float* imgOut = new float[outSize];
    float* imgDownConv =  new float[outSize];
    float* imgDownConvGPU = new float[outSize];
    float* f = new float[outSize];

    for(int i=0; i<inSize; i++)
    {
		dummyGradU[i] = i * 1.0f;
		if(i == 32)
		{
			dummyGradU[i] = 3000.0f;
		}
        imgIn[i] = 1.0f;
    }
    
    for(int i=0; i<kernelSize; i++)
    {
        kernel[i] = 1.0f;
    }    
    
    for(int i=0; i<outSize ; i++)
    {
        f[i] = 3.2f;
        imgOut[i] = 0.0f;
    }
	std::cout<< "outsize: " << outSize << "\n"; 
    // GPU SECTION

    // Initialize GPU arrays
    float* d_imgIn = NULL;
    float* d_imgOut = NULL;
    float* d_kernel = NULL;
    float* d_imgDownConv = NULL;
    float* d_dummyGradU = NULL;
    float* d_f = NULL;
	float* d_eps = NULL;	

    // Allocate memory on the GPU:
    hipMalloc(&d_imgIn, inSize*sizeof(float));
    hipMalloc(&d_imgOut, outSize*sizeof(float));
    hipMalloc(&d_kernel, kernelSize*sizeof(float));
    hipMalloc(&d_imgDownConv, outSize*sizeof(float));
    hipMalloc(&d_dummyGradU, inSize*sizeof(float));
    hipMalloc(&d_f, outSize*sizeof(float));
    hipMalloc(&d_eps, sizeof(float));

    
    // Copy arrays from CPU to GPU:
    hipMemcpy(d_imgIn, imgIn, inSize*sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_kernel, kernel, kernelSize*sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_f, f, outSize*sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_dummyGradU, dummyGradU, inSize*sizeof(float), hipMemcpyHostToDevice);
    
    // Compute downconvolution:
    computeDownConvolutionGlobalMemCuda(d_imgDownConv, d_imgIn, d_kernel, n, m, nc, w, h);

    // Subtract using CUBLAS:
	const float alpha = -1.0f;
	//const float alpha1 = -1.0f;
	//*alpha = alpha1;

    hipblasSaxpy(handle, n*m*nc, &alpha, d_f, 1, d_imgDownConv, 1);
	//hipblasSaxpy(handle, n, alpha, x, 1, y, 1);
    // Calculate epsilon using CUBLAS
    //eps = computeEpsilonCuda(handle, d_imgDownConv, d_dummyGradU, outSize, 5e-3);
    
	computeEpsilonGlobalMemCuda(d_eps, handle, d_imgIn, d_dummyGradU, inSize, 5e-3);
    
    // Copy results back to CPU for visualization:
    hipMemcpy(imgDownConvGPU, d_imgDownConv, outSize*sizeof(float), hipMemcpyDeviceToHost);
    hipMemcpy(&eps, d_eps, sizeof(float), hipMemcpyDeviceToHost);
	
    // CPU commands

    computeDownConvolutionCPU(imgDownConv, imgIn, kernel, n, m, nc, w, h);

    epsCPU = computeEpsilon(imgIn, dummyGradU, inSize, 0.005);

    subtractArrays(imgOut,imgDownConv, f, outSize);

    
    std::cout<<"Input Image"<<std::endl;
    for(int c=0; c<nc; ++c){
        std::cout<<"Channel no :  "<<c<<std::endl;
        for(int j=0; j<m; ++j){
            for(int i=0; i<n; ++i){
                std::cout<<imgIn[i + (j*w) + (c*w*h)]<<"   ";
            }
            std::cout<<std::endl;
        }
    }

    std::cout<<"After CPU Down convolution"<<std::endl;
    for(int c=0; c<nc; ++c){
        std::cout<<"Channel no :  "<<c<<std::endl;
        for(int j=0; j<outSizeY; ++j){
            for(int i=0; i<outSizeX; ++i){
                std::cout<<imgDownConv[i + (j*outSizeX) + (c*outSizeX*outSizeY)]<<"   ";
            }
            std::cout<<std::endl;
        }
    }

    std::cout<<"After CPU subtraction"<< "\n";
    for(int c=0; c<nc; ++c){
        std::cout<<"Channel no :  "<<c<<std::endl;
        for(int j=0; j<outSizeY; ++j){
            for(int i=0; i<outSizeX; ++i){
                std::cout<<imgOut[i + (j*outSizeX) + (c*outSizeX*outSizeY)]<<"   ";
            }
            std::cout<<std::endl;
        }
    }

    std::cout<<"GPU Deconvolution and Subtraction"<< "\n";
    for(int c=0; c<nc; ++c){
        std::cout<<"Channel no :  "<<c<<std::endl;
        for(int j=0; j<outSizeY; ++j){
            for(int i=0; i<outSizeX; ++i){
                std::cout<<imgDownConvGPU[i + (j*outSizeX) + (c*outSizeX*outSizeY)]<<"   ";
            }
            std::cout<<std::endl;
        }
    }

    std::cout<< "\n" << "GPU epsilon value: " << eps << "\n";
    
    std::cout<< "\n" << "CPU epsilon value: " << epsCPU << "\n";

    //Free Cuda Memory
    hipFree(d_imgIn);
    hipFree(d_imgOut);
    hipFree(d_kernel);
    hipFree(d_imgDownConv);
    hipFree(d_dummyGradU);
	hipFree(d_f);
	hipFree(d_eps);


    //Free Host Memory
    delete[] imgIn;
    delete[] imgOut;
    delete[] imgDownConv;
    delete[] kernel;
    delete[] dummyGradU;
    delete[] f;
    delete[] imgDownConvGPU;
    
    hipblasDestroy(handle);
}
