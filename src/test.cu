#include "test.cuh"

int simpleTest(int argc, char** argv){
    //if(argc != 8){
        //std::cout<<"not enough entries"<<std::endl;
        //return -1;
    //}
    size_t nc = 3;
    size_t w = 7; size_t h = 7;
    size_t m = 5; size_t n = 5;
    //size_t w = std::stoi(argv[1]); size_t h = std::stoi(argv[2]);
    //size_t m = std::stoi(argv[4]); size_t n = std::stoi(argv[5]);
    size_t outSizeX = m + w - 1;  size_t outSizeY = n + h - 1;
    size_t inSize = w * h * nc;
    size_t outSize = outSizeX * outSizeY * nc;
    size_t outSizeXdc = m;    size_t outSizeYdc = n;
    size_t outSizedc = outSizeXdc * outSizeYdc * nc;
    float* imgIn = new float[inSize];
    float* imgOut = new float[outSize];
    float* imgUpConv =  new float[outSize];
    float* kernel = new float[m*n];
    float* imgOutDownConv = new float[outSizeXdc * outSizeYdc * nc];
    float* imgBuffer = new float[inSize];
    for(int i=0; i<inSize; ++i)
        imgIn[i] = 1.0f;
    for(int i=0; i<m*n; ++i)
        kernel[i] = 1.0f;

    //CPU commands
    /*padImgCPU(imgOut, imgIn, w, h, nc, m, n);*/
    /*std::cout<<"padding done"<<std::endl;*/
    /*computeUpConvolutionCPU(imgUpConv, imgIn, kernel, w, h, nc, m, n);*/

    //GPU commnds
    float* d_imgIn = NULL;
    float* d_imgOut = NULL;
    float* d_kernel = NULL;
    float* d_imgUpConv = NULL;
    float* d_imgDownConv = NULL;
    float* d_imgInBuffer = NULL;
    hipMalloc(&d_imgIn, inSize*sizeof(float));
    hipMalloc(&d_imgInBuffer, inSize*sizeof(float));
    hipMalloc(&d_imgOut, outSize*sizeof(float));
    hipMalloc(&d_kernel, m*n*sizeof(float));
    hipMalloc(&d_imgUpConv, outSize*sizeof(float));
    hipMalloc(&d_imgDownConv, outSizedc*sizeof(float));
    hipMemcpy(d_imgIn, imgIn, inSize*sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_kernel, kernel, m*n*sizeof(float), hipMemcpyHostToDevice);
    padImgGlobalMemCuda(d_imgOut, d_imgIn, w, h, nc, m, n);
    computeUpConvolutionGlobalMemCuda(d_imgUpConv, d_imgIn, d_kernel, w, h, nc, m, n);
    computeImageConvolution(d_imgDownConv, m, n,
                            d_imgIn, d_imgInBuffer, 
                            w, h, 
                            d_imgOut, outSizeX, outSizeY, 
                            nc); 
    hipMemcpy(imgOut, d_imgOut, outSize*sizeof(float), hipMemcpyDeviceToHost);
    hipMemcpy(imgUpConv, d_imgUpConv, outSize*sizeof(float), hipMemcpyDeviceToHost);
    hipMemcpy(imgOutDownConv, d_imgDownConv, outSizedc*sizeof(float), hipMemcpyDeviceToHost);
    hipMemcpy(imgBuffer, d_imgInBuffer, inSize*sizeof(float), hipMemcpyDeviceToHost);

    std::cout<<"Input Image"<<std::endl;
    for(int c=0; c<nc; ++c){
        std::cout<<"Channel no :  "<<c<<std::endl;
        for(int j=0; j<h; ++j){
            for(int i=0; i<w; ++i){
                std::cout<<imgIn[i + (j*w) + (c*w*h)]<<"   ";
            }
            std::cout<<std::endl;
        }
    }

    std::cout<<"Output Image after padding"<<std::endl;
    for(int c=0; c<nc; ++c){
        std::cout<<"Channel no :  "<<c<<std::endl;
        for(int j=0; j<outSizeY; ++j){
            for(int i=0; i<outSizeX; ++i){
                std::cout<<imgOut[i + (j*outSizeX) + (c*outSizeX*outSizeY)]<<"   ";
            }
            std::cout<<std::endl;
        }
    }

    std::cout<<"After Up convolution"<<std::endl;
    for(int c=0; c<nc; ++c){
        std::cout<<"Channel no :  "<<c<<std::endl;
        for(int j=0; j<outSizeY; ++j){
            for(int i=0; i<outSizeX; ++i){
                std::cout<<imgUpConv[i + (j*outSizeX) + (c*outSizeX*outSizeY)]<<"   ";
            }
            std::cout<<std::endl;
        }
    }

    std::cout << "After Ravil's Down Conv" << std::endl;
    for(int j=0; j<n; ++j){
        for(int i=0; i<m; ++i){
            std::cout<<imgOutDownConv[i + (j*m)]<<"   ";
        }
        std::cout<<std::endl;
    }

    std::cout<<"Buffer"<<std::endl;
    for(int c=0; c<nc; ++c){
        std::cout<<"Channel no :  "<<c<<std::endl;
        for(int j=0; j<h; ++j){
            for(int i=0; i<w; ++i){
                std::cout<<imgBuffer[i + (j*w) + (c*w*h)]<<"   ";
            }
            std::cout<<std::endl;
        }
    }
    //Free Cuda Memory
    hipFree(d_imgIn);
    hipFree(d_imgOut);
    hipFree(d_kernel);
    hipFree(d_imgUpConv);
    hipFree(d_imgDownConv);
    hipFree(d_imgInBuffer);

    //Free Host Memory
    delete[] imgIn;
    delete[] imgOut;
    delete[] imgUpConv;
    delete[] kernel;
    delete[] imgOutDownConv;
    delete[] imgBuffer;
    
}
