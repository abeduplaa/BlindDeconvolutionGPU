#include "hip/hip_runtime.h"
#include <iostream>
#include <hip/hip_runtime.h>
#include <cmath>
#include <algorithm>

#include "epsilon.cuh"
#include "helper.cuh"


float computeEpsilonCuda(hipblasHandle_t handle, const float *a, const float *grad, const int size, const float smallnum)
{
    //initialize indices:
    int a_i = NULL;
    int grad_i = NULL;
    
    // call cublas functions:
    absMaxIdCUBLAS(handle, size, a, 1, &a_i);

    absMaxIdCUBLAS(handle, size, grad, 1, &grad_i);


    // calling cuda kernel
    //return (smallnum * a[a_i]) * ( ( grad[grad_i] < 1e31) ? (1.0/grad[grad_i]) : (1e-31) );
	return (float)grad_i;
}

// CPU FUNCTIONS

float computeMaxElem(const float *array,const float size)
{
    float maxElem = array[0];
	
	for(int i=1; i<size; i++)
	{
		if (array[i]>array[i-1])
		{
			maxElem = array[i];
		}
	}

	//maxElem = *std::max_element(array,array+size);

    return maxElem;
}

//void computeAbsArray(float *absArray, const float *array, size_t size)
void computeAbsArray(float *absarray, const float *array, size_t size)
{
    for(int i = 0; i<size; i++)
    {
        //absArray[i] = fabs(array[i]);
        absarray[i] = fabs(array[i]);        
    }
}

float computeEpsilon(const float *imgIn, const float *gradU, const int size, const float smallnum)
{
    // 5e-3*max(u(:))/max(1e-31,max(max(abs(gradu(:)))));

    // float *absElemG;
    float maxGrad = 0;
    float maxElemU = 0;
    float lower = 0;
    float eps = 0;
	float *absgradU = new float[size];

    computeAbsArray(absgradU, gradU, size);
    
    maxGrad = computeMaxElem(absgradU, size);

    if(1e-31 > maxGrad)
    {
        lower = 1e31;
    }else
    {
        lower = 1/maxGrad;
    }

    maxElemU = computeMaxElem(imgIn, size);

    eps = (smallnum * maxElemU ) * lower;

    return eps;

}
