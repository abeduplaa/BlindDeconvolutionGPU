#include "hip/hip_runtime.h"
#include <iostream>
#include <hip/hip_runtime.h>
#include <cmath>
#include <algorithm>

#include "epsilon.cuh"
#include "helper.cuh"



__global__
void computeEpsilonGlobalMemKernel(float *eps, const float *a, const int a_i, const float *grad, const int grad_i, const float smallnum)
{
	// int idx = threadIdx.x + blockIdx.x*blockDim.x;
	// int idy = threadIdx.y + blockIdx.y*blockDim.y;
    // calling cuda kernel
    *eps = (smallnum * a[a_i]) * ( ( grad[grad_i] < 1e31) ? (1.0/grad[grad_i]) : (1e-31) );
    
}


void computeEpsilonGlobalMemCuda(float *eps, hipblasHandle_t handle, const float *a, const float *grad, const int size, const float smallnum)
{
    	// allocate block and grid size
	dim3 block(32, 8, 1);
	//dim3 grid = computeGrid2D(block, m - h + 1, n - w + 1);
	dim3 grid = computeGrid2D(block, 8, 8);

    //initialize indices:
    int a_i = NULL;
    int grad_i = NULL;
    
    // call cublas functions:
    absMaxIdCUBLAS(handle, size, a, 1, &a_i);

    absMaxIdCUBLAS(handle, size, grad, 1, &grad_i);
	
	//calling cuda kernel
    computeEpsilonGlobalMemKernel <<<grid,block>>> (eps, a, a_i, grad, grad_i, smallnum);
}



// CPU FUNCTIONS

float computeMaxElem(const float *array,const float size)
{
    float maxElem = array[0];
	
	for(int i=1; i<size; i++)
	{
		if (array[i]>array[i-1])
		{
			maxElem = array[i];
		}
	}

	//maxElem = *std::max_element(array,array+size);

    return maxElem;
}

//void computeAbsArray(float *absArray, const float *array, size_t size)
void computeAbsArray(float *absarray, const float *array, size_t size)
{
    for(int i = 0; i<size; i++)
    {
        //absArray[i] = fabs(array[i]);
        absarray[i] = fabs(array[i]);        
    }
}

float computeEpsilon(const float *imgIn, const float *gradU, const int size, const float smallnum)
{
    // 5e-3*max(u(:))/max(1e-31,max(max(abs(gradu(:)))));

    // float *absElemG;
    float maxGrad = 0;
    float maxElemU = 0;
    float lower = 0;
    float eps = 0;
	float *absgradU = new float[size];

    computeAbsArray(absgradU, gradU, size);
    
    maxGrad = computeMaxElem(absgradU, size);

    if(1e-31 > maxGrad)
    {
        lower = 1e31;
    }else
    {
        lower = 1/maxGrad;
    }

    maxElemU = computeMaxElem(imgIn, size);

    eps = (smallnum * maxElemU ) * lower;

    return eps;

}

float computeEpsilonCuda(hipblasHandle_t handle, const float *a, const float *grad, const int size, const float smallnum)
{
    //initialize indices:
    int a_i = NULL;
    int grad_i = NULL;
    
    // call cublas functions:
    absMaxIdCUBLAS(handle, size, a, 1, &a_i);

    absMaxIdCUBLAS(handle, size, grad, 1, &grad_i);


    // calling cuda kernel
    //return (smallnum * a[a_i]) * ( ( grad[grad_i] < 1e31) ? (1.0/grad[grad_i]) : (1e-31) );
	return (float)grad_i;
}
