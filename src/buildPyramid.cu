#include "hip/hip_runtime.h"
#include <iostream>
#include <vector>
#include <math.h>

#include "buildPyramid.cuh"


/////////////////

// GPU FUNCTIONS:
__global__ 
void resizeCudaKernel( unsigned char* input,
    unsigned char* output,
    const int outputWidth,
    const int outputHeight,
    const int inputWidthStep,
    const int outputWidthStep,
    const float pixelGroupSizeX,
    const float pixelGroupSizeY,
    const int inputChannels)
{
//2D Index of current thread
const int outputXIndex = blockIdx.x * blockDim.x + threadIdx.x;
const int outputYIndex = blockIdx.y * blockDim.y + threadIdx.y;

//Only valid threads perform memory I/O
if((outputXIndex<outputWidth) && (outputYIndex<outputHeight))
{
    // Starting location of current pixel in output
    int output_tid  = outputYIndex * outputWidthStep + (outputXIndex * inputChannels);

    // Compute the size of the area of pixels to be resized to a single pixel
    const float pixelGroupArea = pixelGroupSizeX * pixelGroupSizeY;

    // Compute the pixel group area in the input image
    const int intputXIndexStart = int(outputXIndex * pixelGroupSizeX);
    const int intputXIndexEnd = int(intputXIndexStart + pixelGroupSizeX);
    const float intputYIndexStart = int(outputYIndex * pixelGroupSizeY);
    const float intputYIndexEnd = int(intputYIndexStart + pixelGroupSizeY);

    if(inputChannels==1) { // grayscale image
        float channelSum = 0;
        for(int intputYIndex=intputYIndexStart; intputYIndex<intputYIndexEnd; ++intputYIndex) {
            for(int intputXIndex=intputXIndexStart; intputXIndex<intputXIndexEnd; ++intputXIndex) {
                int input_tid = intputYIndex * inputWidthStep + intputXIndex;
                channelSum += input[input_tid];
            }
        }
        output[output_tid] = static_cast<unsigned char>(channelSum / pixelGroupArea);
    } else if(inputChannels==3) { // RGB image
        float channel1stSum = 0;
        float channel2stSum = 0;
        float channel3stSum = 0;
        for(int intputYIndex=intputYIndexStart; intputYIndex<intputYIndexEnd; ++intputYIndex) {
            for(int intputXIndex=intputXIndexStart; intputXIndex<intputXIndexEnd; ++intputXIndex) {
                // Starting location of current pixel in input
                int input_tid = intputYIndex * inputWidthStep + intputXIndex * inputChannels;
                channel1stSum += input[input_tid];
                channel2stSum += input[input_tid+1];
                channel3stSum += input[input_tid+2];
            }
        }
        output[output_tid] = static_cast<unsigned char>(channel1stSum / pixelGroupArea);
        output[output_tid+1] = static_cast<unsigned char>(channel2stSum / pixelGroupArea);
        output[output_tid+2] = static_cast<unsigned char>(channel3stSum / pixelGroupArea);
    } else { 
    }
}
}


void resizeGlobalMemCuda(const float *output, const float *input)
{
    // allocate block and grid size
	dim3 block(32, 8, 1);
    dim3 grid = computeGrid2D(block, w - m + 1, h - n + 1);
    
	//Specify a reasonable block size
	// const dim3 block(16,16);
 
	//Calculate grid size to cover the whole image
	// const dim3 grid((output.cols + block.x - 1)/block.x, (output.rows + block.y - 1)/block.y);
 
	// Calculate how many pixels in the input image will be merged into one pixel in the output image
	const float pixelGroupSizeY = float(input.rows) / float(output.rows);
	const float pixelGroupSizeX = float(input.cols) / float(output.cols);
 
	//Launch the size conversion kernel
	resizeCudaKernel<<<grid,block>>>(d_input,d_output,output.cols,output.rows,input.step,output.step, pixelGroupSizeX, pixelGroupSizeY, input.channels());
 
	timer.Stop();
	printf("Own Cuda code ran in: %f msecs.\n", timer.Elapsed());
 
	//Synchronize to check for any kernel launch errors
	SAFE_CALL(hipDeviceSynchronize(),"Kernel Launch Failed");
 
	//Copy back data from destination device meory to OpenCV output image
	SAFE_CALL(hipMemcpy(output.ptr(),d_output,outputBytes,hipMemcpyDeviceToHost),"CUDA Memcpy Host To Device Failed");
 
	//Free the device memory
	SAFE_CALL(hipFree(d_input),"CUDA Free Failed");
	SAFE_CALL(hipFree(d_output),"CUDA Free Failed");
	//SAFE_CALL(hipDeviceReset(),"CUDA Device Reset Failed");
}


///////
//CPU FUNCTIONS


int kernelDim(int in, const float scaleMultiplier, const int smallestScale)
{
    // calculate out
    int out = round( (float)in / scaleMultiplier );
    
    // check if dimension is even
    out = (out%2 == 0) ? (out-1) : out;

    // check if dimension same as last dimension
    out = (out == in) ? (out-2) : out;

    // check if dimension is  smaller than smallest value
    out = (out < smallestScale) ? smallestScale : out;
    
    return out;
}

int imageDim(int in, const float factor)
{
    int out = round( (float)in / factor );

    // check if dimension is even
    out = (out%2 == 0) ? (out-1) : out;

    return out;

}

int pyramidScale(const int m, const int n, const float smallestScale, 
    const float scaleMultiplier, const float lambdaMultiplier)
{
    //dummy vars:
    int m1 = m;
    int n1 = n;
    int l1 = 0;
    int pyramidSize = 0;

    while( (m1 > smallestScale) && (n1 > smallestScale) 
    && (l1 * lambdaMultiplier < largestLambda) )
    {
        m1 = kernelDim(m1, scaleMultiplier, smallestScale);

        n1 = kernelDim(n1, scaleMultiplier, smallestScale);

        l1 = l1 * lambdaMultiplier;

        pyramidSize += 1;
    }

    return pyramidSize;
}

void buildPyramid(int *wP, int *hP, int *mP, int *nP, float *lambdas, 
    const int w, const int h, const int m, const int n, 
    const int smallestScale, const float lambda, const float finalLambda, 
    const float scaleMultiplier, const float lambdaMultiplier, const int pyramidSize)
{
    float factorW = 0;
    float factorH = 0;

    wP[0] = w;
    hP[0] = h;
    mP[0] = m;
    nP[0] = n;
    lambdas[0] = lambda;
    
    // while( (mP.back() > smallestScale) && (np.back() > smallestScale) 
    // &&  (lambdas.back() * lambdaMultiplier < largestLambda) )
    
    for(int i = 1 ; i < pyramidSize ; i++)
    {
        lambdas[i] = lambdas[i-1] *lambdaMultiplier;

        mP[i] = kernelDim(mP[i - 1], scaleMultiplier, smallestScale);
        nP[i] = kernelDim(nP[i - 1], scaleMultiplier, smallestScale);
        
        factorW = mP[i - 1] / mP[i];
        factorH = nP[i - 1] / nP[i];

        wP[i] = imageDim(wP[i-1] , factorW);
        hP[i] = imageDim(hP[i-1] , factorH);
    }
}



