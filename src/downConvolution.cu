#include "hip/hip_runtime.h"
#include <iostream>
#include <hip/hip_runtime.h>

#include "downConvolution.cuh"
#include "helper.cuh"
#include "../cub-1.8.0/hipcub/hipcub.hpp"

__global__
void computeDownConvolutionGlobalMemKernel(float* imgOut, const float* imgIn,
const float* kernel, const int w, const int h, const int nc, const int m, const int n){
	
	//0. define idxx and idyy in 2d grid
	int idx = threadIdx.x + blockIdx.x*blockDim.x;
	int idy = threadIdx.y + blockIdx.y*blockDim.y;
	
	//1. define necessary variables
	size_t imgOut_w = w - m + 1;
	size_t imgOut_h = h - n + 1;
	int kRadius_m = (m - 1) / 2;
	int kRadius_n = (n - 1) / 2;
	int kidx = 0;

	// GPU Parameters
	int i = idx + idy*imgOut_w;
	int out_idx, out_x, out_y = 0;
	int in_idx = 0;

	//2. compute downconvolution

	//if(idx < imgOut_w && idy < imgOut_h) //why is this incorrect?
	if(idx < imgOut_w && idy < imgOut_h )
	{
		for(int c = 0; c < nc; c++)
		{
            out_idx = i + (c*imgOut_h*imgOut_w);
            out_x = idx + kRadius_m;
            out_y = idy + kRadius_n;
            imgOut[out_idx] = 0.0f;
			
            for(int kj = -kRadius_n; kj <= kRadius_n; kj++)
            {
                for(int ki = -kRadius_m; ki <= kRadius_m; ki++)
                {
                    imgOut[out_idx] += kernel[(ki+kRadius_m)+((kj+kRadius_n)*kRadius_m)]
                        * imgIn[(out_x+ki)+ (out_y+kj)*w + (c*w*h)];
                }
            }
		}
	}
}



void computeDownConvolutionCPU(float *imgOut, const float *imgIn, const float *kernel, int n, int m, int nc, int w, int h)
{
	//0. check if kernel dimensions are less than image dimensions ( h>m+1 or also w>n+1)
	if( (h>m+1) || (w>n+1) )
	{
		std::cout << "\n" << "n: " << n << ", w: " << w << ", m: " << m << ", h: " << h << "\n";
		throw std::invalid_argument("Kernel dimensions are too big!");
	}


	//1. define necessary variables
	int out_idx = 0;
	int in_idx = 0;
	int kidx = 0;
	size_t imgOut_h = m - h + 1;
	size_t imgOut_w = n - w + 1;
	int kRadius_w = (w-1)/2;
	int kRadius_h = (h-1)/2;

	//2. compute downconvolution

	for(int c = 0; c < nc; c++)
	{
		for(int j = 0 ; j < imgOut_h ; j++)
		{
			for(int i = 0 ; i < imgOut_w ; i++)
			{
				out_idx = i + j*imgOut_w + c*imgOut_h*imgOut_w;
				in_idx = out_idx + (kRadius_h*n) + kRadius_w + (kRadius_w*2)*j;
				
				imgOut[out_idx] = 0.0f;
				// std::cout<< "output idx: " << out_idx << ", input idx: " << in_idx << "\n"; 
				
				for(int kj = 0; kj < h; ++kj)
				{
					for(int ki = 0; ki < w; ++ki)
					{
						kidx = in_idx - (kRadius_w - ki) - ( (kRadius_h - kj)*n);
						// std::cout<< "convolution multiplication, k: " << ki+kj*w << ", input: " << kidx << " , value: " << kernel[ki + kj*w] * imgIn[kidx] <<  "\n";
						imgOut[out_idx - c*imgOut_h*imgOut_w] += kernel[ki + kj*w + c*imgOut_h*imgOut_w] * imgIn[kidx];
					}
				}
			}
		} 
	}
}

void computeDownConvolutionGlobalMemCuda(float *imgOut, const float *imgIn, const float *kernel,
        const int w, const int h, const int nc, const int m, const int n)
{

	// allocate block and grid size
	dim3 block(32, 8, 1);
	dim3 grid = computeGrid2D(block, w - m + 1, h - n + 1);

	//calling cuda kernel
	computeDownConvolutionGlobalMemKernel <<<grid,block>>> (imgOut, imgIn, kernel,
											w, h, nc, m, n);
}


// -------------------------------------------------------------------------------------
__global__
void computeImageConvolution(float *imgInBuffer,
                             const float *imgIn,
                             const int w, const int h,
                             const float *imgInPad,
                             const int padw, const int padh,
                             const int delta_x, const int delta_y,
                             const int nc) {

    int idx = threadIdx.x + blockIdx.x * blockDim.x;
    int idy = threadIdx.y + blockIdx.y * blockDim.y;

    for (int channel = 0; channel < nc; ++channel) {

        int offset = w * h * channel;
        int offset_pad_img = padw * padh * channel;

        for (int y = idy; y < h; y += blockDim.y * gridDim.y) {
            for (int x = idx; x < w; x += blockDim.x * gridDim.x) {

                int relative_x = x + delta_x; 
                int relative_y = y + delta_y; 
                
                int kernel_index = getIndex(x, y, w) + offset;
                int image_index = getIndex(relative_x, relative_y, padw) + offset_pad_img;

                imgInBuffer[kernel_index] = imgIn[kernel_index] * imgInPad[image_index];
            }
        }
    }

}


void computeImageConvolution(float *d_kernel_temp, const int mk, const int nk ,
                             const float *d_imgIn, float *d_imgInBuffer, 
                             const int w, const int h, 
                             const float *d_imgInPad, const int padw, const int padh, 
                             const int nc) {

	// allocate block and grid size
	dim3 block(32, 8, 1);
	dim3 grid = computeGrid2D(block, w, h);
    // determine the size of temp buffer
    void *d_temp_storage = NULL;
    size_t temp_storage_bytes = 0;

    // NOTE: When d_temp_storage is NULL, no work is done and
    // the required allocation size is returned in
    // temp_storage_bytes.
    hipcub::DeviceReduce::Sum<float*, float*>(d_temp_storage,
                                           temp_storage_bytes, 
                                           d_imgInBuffer, 
                                           &d_kernel_temp[0], 
                                           w * h * nc);

    hipMalloc(&d_temp_storage, temp_storage_bytes); CUDA_CHECK;

    for(int delta_y = 0; delta_y < nk; ++delta_y){
        for(int delta_x = 0; delta_x < mk; ++delta_x){


            int kernel_index = getIndex(delta_x, delta_y, mk);
            computeImageConvolution<<<grid, block>>>(d_imgInBuffer,
                                                     d_imgIn,
                                                     w, h,
                                                     d_imgInPad,
                                                     padw, padh,
                                                     delta_x, delta_y,
                                                     nc);
            
            CUDA_CHECK;
            hipDeviceSynchronize();


            hipcub::DeviceReduce::Sum<float*, float*>(d_temp_storage,
                                                   temp_storage_bytes,
                                                   d_imgInBuffer,
                                                   &d_kernel_temp[kernel_index],
                                                   w * h * nc);
            hipDeviceSynchronize();
        }
    }
    hipFree(d_temp_storage);  CUDA_CHECK;
}



