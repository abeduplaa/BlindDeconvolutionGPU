// ########################################################################
// Practical Course: GPU Programming in Computer Vision
// Technical University of Munich, Computer Vision Group
// ########################################################################

#include <iostream>
#include <iomanip>
#include <string>
#include <opencv2/core/core.hpp>
#include <opencv2/highgui/highgui.hpp>
#include <opencv2/imgproc/imgproc.hpp>

#include "test.cuh"
#include "helper.cuh"
#include "downConvolution.cuh"
#include "testdownConv.cuh"
#include "divergence.cuh"
#include "adjustImage.cuh"
#include "pad.cuh"
#include "upConvolution.cuh"
#include "epsilon.cuh"
#include "selectNonZero.cuh"
#include "normalise.cuh"

#include "hipblas.h"


/*int main(int argc,char **argv)*/
/*{*/
    /*downConvTest();*/
/*}*/



 int main(int argc,char **argv) {

    // TODO: ADD COMMAND LINE FUNCTIONS LATER

    // parse command line parameters
    const char *params = {
        "{image| |input image}"
        "{bw|false|load input image as grayscale/black-white}"
        "{mk|5|kernel width }"
        "{nk|5|kernel height}"
        "{cpu|false|compute on CPU}"
        "{eps|1e-3| epsilon }"
        "{lambda|0.0068| lambda }"
        "{iter|1| iter}"
       // "{m|mem|0|memory: 0=global, 1=shared, 2=texture}"
    };
    cv::CommandLineParser cmd(argc, argv, params);

    // input image
    std::string inputImage = cmd.get<std::string>("image");
    // number of computation repetitions to get a better run time measurement
    // size_t repeats = (size_t)cmd.get<int>("repeats");
    // load the input image as grayscale
     bool gray = cmd.get<bool>("bw");
	 int mk = cmd.get<int>("mk"); mk = (mk <= 0) ? 5 : mk;
	 int nk = cmd.get<int>("nk"); nk = (nk <= 0) ? 5 : nk;
     bool is_cpu = cmd.get<bool>("cpu");
     float lambda = cmd.get<float>("lambda"); lambda = (lambda <= 0) ? 0.0068 : lambda; 
     float lambda_min = 0.0006f;
     float eps = cmd.get<float>("eps"); eps = ( eps <= 0 ) ? 1e-3 : eps;
     int iter = cmd.get<int>("iter"); iter = ( iter <= 0 ) ? 1 : iter;


     std::cout << "mode: " << (is_cpu ? "CPU" : "GPU") << std::endl;

    // TODO: LOAD IMAGE
    // read input frame
    cv::Mat mIn;
    // load the input image using opencv (load as grayscale if "gray==true", 
    // otherwise as is (may be color or grayscale))
    mIn = cv::imread(inputImage.c_str(), (gray ? CV_LOAD_IMAGE_GRAYSCALE : -1));

    // check
    if (mIn.empty()) {
        std::cerr << "ERROR: Could not retrieve frame " << inputImage << std::endl;
        return 1;
    }
    // convert to float representation (opencv loads image values as single bytes by default)
    mIn.convertTo(mIn, CV_32F);

    // get image dimensions
    int w = mIn.cols;         // width
    int h = mIn.rows;         // height
    int nc = mIn.channels();  // number of channels
    std::cout << "Original Image: " << w << " x " << h  << " x " << nc << std::endl;
    adjustImageSizeToOdd(mIn, w, h, nc);

	size_t img_size = w * h * nc;
    size_t padw = w + mk - 1;
    size_t padh = h + nk - 1;
    size_t pad_img_size = padw * padh * nc;
    std::cout << "Image after Cropping: " << w << " x " << h  << " x " << nc << std::endl;
    std::cout << "Pad Image size" << padw << " x " << padh << " x " << nc << std::endl;

    // init kernel
    size_t kn = mk*nk;
    float *kernel = new float[kn * sizeof(float)];
    //  initialize kernel to uniform.
    initialiseKernel(kernel, mk, nk);

    // initialize CUDA context
    // hipDeviceSynchronize();

    // Initialize Cublas
    hipblasHandle_t handle;
    hipblasCreate(&handle);

    // ### Set the output image format
    cv::Mat mOut(h, w, mIn.type());  // grayscale or color depending on input image, nc layers

    // ### Allocate arrays
    float *imgIn = new float[img_size];
    float *imgInPad = new float[pad_img_size]; //TODO: check size (RAVI)
    float *imgOut = new float[pad_img_size];
    float *dx_fw = new float[pad_img_size];
    float *dy_fw = new float[pad_img_size];
    float *dx_bw = new float[pad_img_size];
    float *dy_bw = new float[pad_img_size];
    float *dx_mixed = new float[pad_img_size];
    float *dy_mixed = new float[pad_img_size];
    float *imgDownConv0 = new float[img_size];
    float *imgUpConv = new float[pad_img_size];

    float *div = new float[pad_img_size];
    float epsU, epsK = 0.0f;
    float alpha = -1.0f;


    // TODO: ALLOCATE MEMORY ON GPU
    // allocate arrays on GPU
    float *d_imgIn = NULL;
    float *d_imgInPad = NULL;
    float *d_imgInBuffer = NULL;
    float *d_imgPadRot = NULL;
    float *d_imgOut = NULL;

    float *d_dx_fw = NULL;
    float *d_dy_fw = NULL;
    float *d_dx_bw = NULL;
    float *d_dy_bw = NULL;
    float *d_dx_mixed = NULL;
    float *d_dy_mixed = NULL;

    float *d_imgDownConv0 = NULL;
    float *d_imgDownConv1 = NULL;

    float *d_imgDownConv1Rot = NULL;

    float *d_imgUpConv = NULL;

    float *d_div = NULL;
    float *d_kernel = NULL;
    float *d_kernel_temp = NULL;

    float *d_epsU = NULL;
    float *d_epsK = NULL;

    hipMalloc(&d_imgIn, img_size * sizeof(float)); CUDA_CHECK;
    hipMalloc(&d_imgInBuffer, img_size * sizeof(float)); CUDA_CHECK;

    // TODO: be sure the size id right (RAVI)
    hipMalloc(&d_imgInPad, pad_img_size * sizeof(float)); CUDA_CHECK;
    hipMalloc(&d_imgPadRot, pad_img_size * sizeof(float)); CUDA_CHECK;
    hipMalloc(&d_imgOut , pad_img_size * sizeof(float)); CUDA_CHECK;

    hipMalloc(&d_kernel, kn * sizeof(float)); CUDA_CHECK;
    hipMalloc(&d_kernel_temp, kn  * sizeof(float)); CUDA_CHECK;

    hipMalloc(&d_dx_fw, pad_img_size * sizeof(float)); CUDA_CHECK;
    hipMalloc(&d_dy_fw, pad_img_size * sizeof(float)); CUDA_CHECK;

    hipMalloc(&d_dx_bw, pad_img_size * sizeof(float)); CUDA_CHECK;
    hipMalloc(&d_dy_bw, pad_img_size * sizeof(float)); CUDA_CHECK;

    hipMalloc(&d_dx_mixed, pad_img_size * sizeof(float)); CUDA_CHECK;
    hipMalloc(&d_dy_mixed, pad_img_size * sizeof(float)); CUDA_CHECK;

    hipMalloc(&d_div , pad_img_size * sizeof(float)); CUDA_CHECK;

    hipMalloc(&d_imgDownConv0, img_size * sizeof(float)); CUDA_CHECK;
    hipMalloc(&d_imgDownConv1, img_size * sizeof(float)); CUDA_CHECK;

    hipMalloc(&d_imgDownConv1Rot, img_size * sizeof(float)); CUDA_CHECK;

    hipMalloc(&d_imgUpConv, pad_img_size * sizeof(float)); CUDA_CHECK;
    hipMalloc(&d_epsU, sizeof(float)); CUDA_CHECK;
    hipMalloc(&d_epsK, sizeof(float)); CUDA_CHECK;

	mIn /= 255.0f;
	convertMatToLayered(imgIn, mIn);
    hipMemcpy(d_imgIn, imgIn, img_size * sizeof(float), hipMemcpyHostToDevice); CUDA_CHECK;
    hipMemcpy(d_kernel, kernel, kn * sizeof(float), hipMemcpyHostToDevice); CUDA_CHECK;

    padImgGlobalMemCuda(d_imgInPad, d_imgIn, w, h, nc, mk, nk);

    for(int iterations = 0; iterations < iter; ++iterations){
        std::cout << "Iteration num:  " << iterations << std::endl;

        // TODO: compute(mirror, rotate) kernel
        rotateKernel_180(d_kernel_temp, d_kernel, mk, nk); 
        hipDeviceSynchronize();

        computeDownConvolutionGlobalMemCuda(d_imgDownConv0, 
                                            d_imgInPad, 
                                            d_kernel_temp, 
                                            padw, 
                                            padh, 
                                            nc, 
                                            mk, 
                                            nk);
        hipDeviceSynchronize();
        /*if(iterations == 1)*/
            /*break;*/
        // DONE: cublas subtract k(+)*u - f. Move that to a separate function
        alpha = -1.0f;
        hipblasSetPointerMode(handle, HIPBLAS_POINTER_MODE_HOST);
        hipblasSaxpy(handle, img_size, &alpha, d_imgIn, 1, d_imgDownConv0, 1); CUDA_CHECK;
        hipDeviceSynchronize();

        // TODO: check the list of  parameters 
        computeUpConvolutionGlobalMemCuda(d_imgUpConv, d_imgDownConv0, d_kernel, w, h, nc, mk, nk);
        hipDeviceSynchronize();

        // compute gradient and divergence
        computeDiffOperatorsCuda(d_div, 
                                 d_dx_fw, d_dy_fw,
                                 d_dx_bw, d_dy_bw,
                                 d_dx_mixed, d_dy_mixed, 
                                 d_imgInPad, padw, padh, nc, 1.0f, eps);
        hipDeviceSynchronize();
        

        // TODO: subtract the divergence from upconvolution result (RAVIL)
        alpha = -1.0f * lambda;

        hipblasSaxpy(handle, pad_img_size, &alpha, d_div, 1, d_imgUpConv, 1); CUDA_CHECK;
        // TODO: compute epsilon on GPU
        computeEpsilonGlobalMemCuda(d_epsU, handle, d_imgInPad, d_imgUpConv, pad_img_size, 5e-3);
        hipDeviceSynchronize();
        

        // TODO: update output image u = u - eps*grad
            // USE CUBLAS AXPY() FUNCTION HERE
        hipblasSetPointerMode(handle, HIPBLAS_POINTER_MODE_DEVICE);
        hipblasSaxpy(handle, pad_img_size, d_epsU, d_imgUpConv, 1, d_imgInPad, 1);
        hipDeviceSynchronize();


        //convoluton of k^y*y^{t+1}
        computeDownConvolutionGlobalMemCuda(d_imgDownConv1, 
                                            d_imgInPad, 
                                            d_kernel_temp, 
                                            padw, 
                                            padh, 
                                            nc, 
                                            mk, nk);
        hipDeviceSynchronize();

        //Substraction with f
        alpha = -1.0f;
        hipblasSetPointerMode(handle, HIPBLAS_POINTER_MODE_HOST);
        hipblasSaxpy(handle, img_size, &alpha, d_imgIn, 1, d_imgDownConv1, 1); CUDA_CHECK;
        hipDeviceSynchronize();

        // flip image
        // Ravi has checked that rotation is correct
        for(int c = 0; c < nc; ++c){
            rotateKernel_180(&d_imgPadRot[c*padw*padh], &d_imgInPad[c*padw*padh], 
                    padw, padh); 
            rotateKernel_180(&d_imgDownConv1Rot[c*w*h], &d_imgDownConv1[c*w*h], w, h);
            hipDeviceSynchronize();
        }

        // TODO: perform convolution: k = u * u_pad
        // computeDownConvolutionGlobalMemCuda1(d_kernel_temp, 
        //                                    d_imgPadRot, 
        //                                    d_imgDownConv1Rot, 
        //                                    padw, 
        //                                    padh, 
        //                                    nc, 
        //                                    w, h);

        computeImageConvolution(d_kernel_temp, mk, nk ,
                                d_imgDownConv1Rot, d_imgInBuffer, 
                                w, h, 
                                d_imgPadRot, padw, padh, 
                                nc); 
        hipDeviceSynchronize();
        
        computeEpsilonGlobalMemCuda(d_epsK, handle, d_kernel, d_kernel_temp, kn, 1e-3);
        hipDeviceSynchronize();
        /*hipMemcpy(kernel, d_kernel_temp, kn*sizeof(float), hipMemcpyDeviceToHost);*/

        /*std::cout << "Grad k" << std::endl;*/
        /*for(int i = 0;  i < nk; ++i){*/
            /*for(int j = 0; j < mk; ++j){*/
                /*std::cout << kernel[j + i*mk] << "   ";*/
            /*}*/
            /*std::cout << std::endl;*/
        /*}*/

        //update kernel
        hipblasSetPointerMode(handle, HIPBLAS_POINTER_MODE_DEVICE);
        hipblasSaxpy(handle, kn, d_epsK, d_kernel_temp, 1, d_kernel, 1);
        hipDeviceSynchronize();

        /*hipMemcpy(kernel, d_kernel, kn*sizeof(float), hipMemcpyDeviceToHost);*/

        /*std::cout << "After time update" << std::endl;*/
        /*for(int i = 0;  i < nk; ++i){*/
            /*for(int j = 0; j < mk; ++j){*/
                /*std::cout << kernel[j + i*mk] << "   ";*/
            /*}*/
            /*std::cout << std::endl;*/
        /*}*/

        //select non zero kernel
        selectNonZeroGlobalMemCuda(d_kernel, mk, nk);
        hipDeviceSynchronize();

        //normalise kernel
        normaliseGlobalMemCuda(d_kernel, mk, nk);
        hipDeviceSynchronize();

        //update lambda
        lambda = 0.99f * lambda;
        if(lambda < lambda_min){
            lambda = lambda_min;
        }

    }
	// convert range of each channel to [0,1]
	// init raw input image array (and convert to layered)

	// TODO:  IMPLEMENT THESE FUNCTIONS

	// 1. pre-process: pad image
	

// 2. perform blind de-convolution



	// padImage(imgIn);
	// computeDeconvolution(imgOut, imgIn, kernel, w, h, nc);


	//hipMemcpy(imgOut,d_imgOut,nbytes,hipMemcpyDeviceToHost);

	// show input image
    
    cv::Mat m_dx(padh, padw, mIn.type());
    cv::Mat m_dy(padh, padw, mIn.type());
    cv::Mat m_div(padh, padw, mIn.type());
    cv::Mat mPadImg(padh, padw, mIn.type());
    cv::Mat mImgDownConv0(h, w, mIn.type());
    cv::Mat mImgUpConv(padh, padw, mIn.type());
    cv::Mat mKernel(nk, mk, 1);
    
    hipMemcpy(&epsU, d_epsU, sizeof(float), hipMemcpyDeviceToHost);
    hipMemcpy(&epsK, d_epsK, sizeof(float), hipMemcpyDeviceToHost);
    hipMemcpy(dx_fw, d_dx_fw, pad_img_size * sizeof(float), hipMemcpyDeviceToHost); CUDA_CHECK;
    hipMemcpy(imgInPad, d_imgInPad, pad_img_size* sizeof(float), hipMemcpyDeviceToHost); CUDA_CHECK;
    hipMemcpy(dy_fw, d_dy_fw, pad_img_size * sizeof(float), hipMemcpyDeviceToHost); CUDA_CHECK;

    hipMemcpy(dx_bw, d_dx_bw, pad_img_size * sizeof(float), hipMemcpyDeviceToHost); CUDA_CHECK;
    hipMemcpy(dy_bw, d_dy_bw, pad_img_size * sizeof(float), hipMemcpyDeviceToHost); CUDA_CHECK;

    hipMemcpy(dx_mixed, d_dx_mixed, pad_img_size * sizeof(float), hipMemcpyDeviceToHost);
    CUDA_CHECK;

hipMemcpy(dy_mixed, d_dy_mixed, pad_img_size * sizeof(float), hipMemcpyDeviceToHost);
    CUDA_CHECK;

    hipMemcpy(imgDownConv0, d_imgDownConv0, img_size * sizeof(float), hipMemcpyDeviceToHost);
    CUDA_CHECK;
    
    hipMemcpy(kernel, d_kernel, kn * sizeof(float), hipMemcpyDeviceToHost);
    CUDA_CHECK;

    hipMemcpy(div, d_div, pad_img_size * sizeof(float), hipMemcpyDeviceToHost); CUDA_CHECK;
    hipMemcpy(imgUpConv, d_imgUpConv, pad_img_size * sizeof(float), hipMemcpyDeviceToHost);
    CUDA_CHECK;

    //copy data from GPU to CPU
    /*float scale = 10.0;*/
    /*for (size_t i = 0; i < pad_img_size; ++i) {*/
        /*dx_fw[i] *= scale;*/
        /*dy_fw[i] *= scale;*/

        /*dx_bw[i] *= scale;*/
        /*dy_bw[i] *= scale;*/

        /*dx_mixed[i] *= scale;*/
        /*dy_mixed[i] *= scale;*/
        /*div[i] *= scale;*/
        /*std::cout << i << "  ---  " << dx_fw[i] << std::endl;*/
    /*}*/
    //DEBUG STARTS
    /*simpleTest(argc, argv);*/
    std::cout << "Value of epsilonU: " << epsU << std::endl;
    std::cout << "Value of epsilonK: " << epsK << std::endl;
    std::cout << "Value of lambda at the end: " << lambda << std::endl;

    for(int i = 0;  i < nk; ++i){
        for(int j = 0; j < mk; ++j){
            std::cout << std::setprecision(7) << kernel[j + i*mk] << "   ";
        }
        std::cout << std::endl;
    }
    std::cout << "YOLO" << std::endl; 
    for(int i = 0;  i < 5; ++i){
        for(int j = 0; j < 5; ++j){
            std::cout << std::setprecision(7) << imgDownConv0[j +  i*w] << "   ";
        }
        std::cout << std::endl;
    }

    //DEBUG ENDS
    

	// show output image: first convert to interleaved opencv format from the layered raw array
    convertLayeredToMat(m_dx, dx_mixed); 
    convertLayeredToMat(m_dy, dy_mixed); 
    convertLayeredToMat(m_div, div);
    convertLayeredToMat(mPadImg, imgInPad);
    convertLayeredToMat(mImgDownConv0, imgDownConv0);
    convertLayeredToMat(mImgUpConv, imgUpConv);

    size_t pos_orig_x = 100, pos_orig_y = 50, shift_y = 50; 
    showImage("Input", mIn, pos_orig_x, pos_orig_y);
    /*showImage("dx", m_dx, pos_orig_x + w, pos_orig_y);*/
    /*showImage("dy", m_dy, pos_orig_x, pos_orig_y + w + shift_y);*/
    showImage("divergence", m_div, pos_orig_x + w, pos_orig_y + w + shift_y);
    showImage("Output Image", mPadImg, 100, 140);
    showImage("Kernel", mKernel, 150, 200);
    /*showImage("Down Conv 0", mImgDownConv0, 200, 240);*/
    /*showImage("Up Conv", mImgUpConv, 300, 340);*/

	//convertLayeredToMat(mOut, imgOut);
	//showImage("Output", mOut, 100+w+40, 100);

    // save results
    /*cv::imwrite("image_input.png",mIn*255.f); */
    /*cv::imwrite("image_result.png",m_div*255.f);*/
    /*cv::imwrite("image_kernel.png",mKernel*255.f);*/

    cv::waitKey(0);

    // Free allocated arrays
    delete [] imgIn;
    delete [] imgInPad; 
    delete [] imgOut;

    delete [] dx_fw;
    delete [] dy_fw;
    delete [] dx_bw;
    delete [] dy_bw;
    delete [] dx_mixed;
    delete [] dy_mixed;

    delete [] imgDownConv0;
    delete [] imgUpConv;

    delete [] div;
    delete [] kernel;

    hipFree(d_imgIn); CUDA_CHECK;
    hipFree(d_imgInPad); CUDA_CHECK;
    hipFree(d_imgPadRot); CUDA_CHECK;
    hipFree(d_imgInBuffer); CUDA_CHECK;
    hipFree(d_imgOut); CUDA_CHECK;
    hipFree(d_kernel); CUDA_CHECK;
    hipFree(d_kernel_temp); CUDA_CHECK;

    hipFree(d_dx_fw); CUDA_CHECK;
    hipFree(d_dy_fw); CUDA_CHECK;
    hipFree(d_dx_bw); CUDA_CHECK;
    hipFree(d_dy_bw); CUDA_CHECK;
    hipFree(d_dx_mixed); CUDA_CHECK;
    hipFree(d_dy_mixed); CUDA_CHECK;

    hipFree(d_imgDownConv0); CUDA_CHECK;
    hipFree(d_imgDownConv1); CUDA_CHECK;
    hipFree(d_imgDownConv1Rot); CUDA_CHECK;
    hipFree(d_imgUpConv); CUDA_CHECK;
    hipFree(d_epsU); CUDA_CHECK;
    hipFree(d_epsK); CUDA_CHECK;

    hipFree(d_div); CUDA_CHECK;

    hipblasDestroy(handle);

    // close all opencv windows
    cv::destroyAllWindows();

    return 0;
} 




