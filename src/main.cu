// ########################################################################
// Practical Course: GPU Programming in Computer Vision
// Technical University of Munich, Computer Vision Group
// ########################################################################

#include <iostream>
#include <iomanip>
#include <string>
#include <opencv2/core/core.hpp>
#include <opencv2/highgui/highgui.hpp>
#include <opencv2/imgproc/imgproc.hpp>

#include "helper.cuh"
#include "downConvolution.cuh"
#include "testdownConv.cuh"
#include "divergence.cuh"
#include "adjustImage.cuh"
#include "pad.cuh"
#include "upConvolution.cuh"
#include "epsilon.cuh"
#include "selectNonZero.cuh"
#include "normalise.cuh"
#include "buildPyramid.cuh"
#include "cudnnDownConvolution.cuh"
#include "cudnnUpConvolution.cuh"

#include <Python.h>
#include <numpy/arrayobject.h>

#include "hipblas.h"

#define INTERPOLATION_METHOD CV_INTER_CUBIC
/*#define INTERPOLATION_METHOD CV_INTER_LINEAR*/

/*int main(int argc,char **argv)*/
/*{*/
    /*downConvTest();*/
/*}*/


void saveMatrixMatlab(const char *key_name,
                      float *array,
                      int dim_x,
                      int dim_y,
                      int dim_z) {

    PyObject *pName, *pModule, *pFunc;
    PyObject *pArgs;

    // load module and function
    /*pName = PyString_FromString("pyfunctions");*/
    pName = PyUnicode_DecodeFSDefault("pyfunctions");
    pModule = PyImport_Import(pName);
    Py_DECREF(pName);

    if (pModule == NULL) {
        std::cout << "ERROR: cannot import scipy IO" << std::endl;
        PyErr_Print();
        exit(1);
    }

    // convert C-array to numpy array
    pFunc = PyObject_GetAttrString(pModule, "save_matrix");
    if (PyCallable_Check(pFunc) == 0) {
        std::cout << "ERROR: cannot link savemat function" << std::endl;
        PyErr_Print();
        exit(1);
    }

    const int num_dims = 3;
    npy_intp dims[num_dims] = {dim_z, dim_y, dim_x};
    PyObject *numpy_array = PyArray_SimpleNewFromData(num_dims,
                                                      dims,
                                                      NPY_FLOAT,
                                                      array);

    // create and init a dictionary to write data to a text file
    PyObject* key = PyUnicode_FromString(key_name); 

    // set up patameters for python function call
    pArgs = PyTuple_New(2);
    PyTuple_SetItem(pArgs, 0, key);
    PyTuple_SetItem(pArgs, 1, numpy_array);

    // call python
    PyObject *pOutput = PyObject_CallObject(pFunc, pArgs);
    if (pOutput == NULL) {
        std::cout << "cannot call save_matrix" << std::endl;
        PyErr_Print();
        exit(1);
    }
}


 int main(int argc,char **argv) {
    Py_Initialize();
    import_array();
    // TODO: ADD COMMAND LINE FUNCTIONS LATER

    // parse command line parameters
    const char *params = {
        "{image| |input image}"
        "{bw|false|load input image as grayscale/black-white}"
        "{cpu|false|compute on CPU}"
        "{mk|5|kernel width }"
        "{nk|5|kernel height}"
        "{eps|1e-3| epsilon }"
        "{lambda|0.00035|lambda }"
        "{iter|10| iter}"
        "{bc|r| bc}"
    };
    cv::CommandLineParser cmd(argc, argv, params);

    // input image
    std::string inputImage = cmd.get<std::string>("image");

    bool gray = cmd.get<bool>("bw");
	int mk = cmd.get<int>("mk"); 
    mk = (mk <= 0) ? 5 : mk;
	int nk = cmd.get<int>("nk"); nk = (nk <= 0) ? 5 : nk;
    bool is_cpu = cmd.get<bool>("cpu");
    float lambda = cmd.get<float>("lambda"); 
    std::cout << "lambda" << lambda << std::endl;
    lambda = (lambda <= 0) ? 0.00035 : lambda; 
    float lambda_min = 0.00001f;
    float eps = cmd.get<float>("eps"); 
    eps = ( eps <= 0 ) ? 1e-3 : eps;
    int iter = cmd.get<int>("iter"); 
    iter = ( iter <= 0 ) ? 10 : iter;
    std::string bc1 = cmd.get<std::string>("bc");   
	char bc = bc1.c_str()[0];
    BoundaryCondition boundary;
    selectBoundaryCondition(bc, boundary);


    std::cout << "mode: " << (is_cpu ? "CPU" : "GPU") << std::endl;

    // TODO: LOAD IMAGE
    // read input frame
    cv::Mat mIn;
    // load the input image using opencv (load as grayscale if "gray==true", 
    // otherwise as is (may be color or grayscale))
    mIn = cv::imread(inputImage.c_str(), (gray ? CV_LOAD_IMAGE_GRAYSCALE : -1));

    // check
    if (mIn.empty()) {
        std::cerr << "ERROR: Could not retrieve frame " << inputImage << std::endl;
        return 1;
    }
    // convert to float representation (opencv loads image values as single bytes by default)
    mIn.convertTo(mIn, CV_32F);

    // get image dimensions
    int w = mIn.cols;         // width
    int h = mIn.rows;         // height
    int nc = mIn.channels();  // number of channels
    std::cout << "Original Image: " << w << " x " << h  << " x " << nc << std::endl;
    adjustImageSizeToOdd(mIn, w, h, nc);

	size_t img_size = w * h * nc;
    size_t padw = w + mk - 1;
    size_t padh = h + nk - 1;
    size_t pad_img_size = padw * padh * nc;
    std::cout << "Image after Cropping: " << w << " x " << h  << " x " << nc << std::endl;
    std::cout << "Pad Image size" << padw << " x " << padh << " x " << nc << std::endl;

    // init kernel
    size_t kn = mk*nk;
    float *kernel = new float[kn * sizeof(float)];
    //  initialize kernel to uniform.
    initialiseKernel(kernel, mk, nk);

    // initialize CUDA context
    // hipDeviceSynchronize();

    // Initialize Cublas
    hipblasHandle_t handle;
    hipblasCreate(&handle);

    //Initialize CUDNN
    hipdnnHandle_t cudnn;
    hipdnnCreate(&cudnn);

    //Pyramid parameter
    int pyramidSize =  0;
    const int smallestKernel = 3;
    const float kernelMultiplier = 1.1f;
    const float lambdaMultiplier = 1.9f;
    const float largestLambda = .11f;
    pyramidSize = pyramidScale(mk,nk, smallestKernel, kernelMultiplier, lambdaMultiplier, lambda, largestLambda);
    std::cout<< "Pyramid Size: " << pyramidSize << std::endl;
    // Build pyramid paramters:
    int *wP = new int[pyramidSize];
    int *hP = new int[pyramidSize];
    int *mP = new int[pyramidSize];
    int *nP = new int[pyramidSize];
    float *lambdas = new float[pyramidSize];

    buildPyramid1(wP, hP, mP, nP, lambdas, w, h, mk, nk, smallestKernel, kernelMultiplier, lambdaMultiplier, lambda, pyramidSize);


	// DEBUGGING:
	std::cout<< "pyramid facts: mP:  ";
	for(int i=0; i<pyramidSize; i++)
	{
		std::cout<< mP[i] << ", ";
	}
	std::cout<< "\n";
    std::cout<< "pyramid facts: nP:  " ;
    for(int i=0; i<pyramidSize; i++)
    {
        std::cout<< nP[i] << ", ";
    }
    std::cout<< "\n";
    std::cout<< "pyramid facts: wP:  " ;
    for(int i=0; i<pyramidSize; i++)
    {
        std::cout<< wP[i] << ", ";
    }
    std::cout<< "\n";
    std::cout<< "pyramid facts: hP:  ";
    for(int i=0; i<pyramidSize; i++)
    {
        std::cout<< hP[i] << ", ";
    }
    std::cout<< "\n";
    std::cout<< "pyramid facts: lambdas:  ";
    for(int i=0; i<pyramidSize; i++)
    {
        std::cout<< lambdas[i] << ", ";
    }
    std::cout<< "\n";


    // ### Set the output image format
    cv::Mat mOut(h, w, mIn.type());  // grayscale or color depending on input image, nc layers

    // ### Allocate arrays
    float *imgIn = new float[img_size];
    float *imgInPad = new float[pad_img_size]; //TODO: check size (RAVI)
    float *imgOut = new float[pad_img_size];
    float *dx_fw = new float[pad_img_size];
    float *dy_fw = new float[pad_img_size];
    float *dx_bw = new float[pad_img_size];
    float *dy_bw = new float[pad_img_size];
    float *dx_mixed = new float[pad_img_size];
    float *dy_mixed = new float[pad_img_size];
    float *imgDownConv0 = new float[img_size];
    float *imgUpConv = new float[pad_img_size];

    float *div = new float[pad_img_size];
    float epsU, epsK = 0.0f;
    float alpha = -1.0f;


    // TODO: ALLOCATE MEMORY ON GPU
    // allocate arrays on GPU
    float *d_imgIn = NULL;
    float *d_imgInPad = NULL;
    float *d_imgInPadTemp = NULL;
    float *d_imgInBuffer = NULL;
    float *d_imgPadRot = NULL;
    float *d_imgOut = NULL;

    float *d_dx_fw = NULL;
    float *d_dy_fw = NULL;
    float *d_dx_bw = NULL;
    float *d_dy_bw = NULL;
    float *d_dx_mixed = NULL;
    float *d_dy_mixed = NULL;

    float *d_imgDownConv0 = NULL;
    float *d_imgDownConv1 = NULL;

    float *d_imgDownConv1Rot = NULL;

    float *d_imgUpConv = NULL;

    float *d_div = NULL;
    float *d_kernel = NULL;
    float *d_kernel_temp = NULL;

    float *d_epsU = NULL;
    float *d_epsK = NULL;

    float *d_comm_buffer = NULL;

    hipMalloc(&d_imgIn, img_size * sizeof(float)); CUDA_CHECK;
    hipMalloc(&d_imgInBuffer, img_size * sizeof(float)); CUDA_CHECK;

    // TODO: be sure the size id right (RAVI)
    hipMalloc(&d_imgInPad, pad_img_size * sizeof(float)); CUDA_CHECK;
    hipMalloc(&d_imgInPadTemp, pad_img_size * sizeof(float)); CUDA_CHECK;
    hipMalloc(&d_imgPadRot, pad_img_size * sizeof(float)); CUDA_CHECK;
    hipMalloc(&d_imgOut , pad_img_size * sizeof(float)); CUDA_CHECK;

    hipMalloc(&d_kernel, kn * sizeof(float)); CUDA_CHECK;
    hipMalloc(&d_kernel_temp, kn  * sizeof(float)); CUDA_CHECK;

    hipMalloc(&d_dx_fw, pad_img_size * sizeof(float)); CUDA_CHECK;
    hipMalloc(&d_dy_fw, pad_img_size * sizeof(float)); CUDA_CHECK;

    hipMalloc(&d_dx_bw, pad_img_size * sizeof(float)); CUDA_CHECK;
    hipMalloc(&d_dy_bw, pad_img_size * sizeof(float)); CUDA_CHECK;

    hipMalloc(&d_dx_mixed, pad_img_size * sizeof(float)); CUDA_CHECK;
    hipMalloc(&d_dy_mixed, pad_img_size * sizeof(float)); CUDA_CHECK;

    hipMalloc(&d_div , pad_img_size * sizeof(float)); CUDA_CHECK;

    hipMalloc(&d_imgDownConv0, img_size * sizeof(float)); CUDA_CHECK;
    hipMalloc(&d_imgDownConv1, img_size * sizeof(float)); CUDA_CHECK;

    hipMalloc(&d_imgDownConv1Rot, img_size * sizeof(float)); CUDA_CHECK;

    hipMalloc(&d_imgUpConv, pad_img_size * sizeof(float)); CUDA_CHECK;
    hipMalloc(&d_epsU, sizeof(float)); CUDA_CHECK;
    hipMalloc(&d_epsK, sizeof(float)); CUDA_CHECK;

    hipMalloc(&d_comm_buffer, pad_img_size * sizeof(float)); CUDA_CHECK;

	mIn /= 255.0f;
	convertMatToLayered(imgIn, mIn);
    hipMemcpy(d_imgIn, imgIn, img_size * sizeof(float), hipMemcpyHostToDevice); CUDA_CHECK;
    hipMemcpy(d_kernel, kernel, kn * sizeof(float), hipMemcpyHostToDevice); CUDA_CHECK;

    padImgGlobalMemCuda(d_imgInPad, 
                        d_imgIn, 
                        w, h, nc, 
                        mk, nk, 
                        boundary);
    //Start loop  for pyramid
    
    int START_LEVEL = pyramidSize - 1;
    /*int START_LEVEL = 0;*/
    std::cout<< "Starting level" << START_LEVEL << std::endl;
    for(int level = START_LEVEL; level >= 0; --level){
		//break;
        //copy image and padded image back to CPU
        std::cout << "Level Number:   " << level << 
            ", lambda: " << lambdas[level] << 
            "mk" << mP[level] << "nk" << nP[level]  <<std::endl;
        hipMemcpy(imgIn, d_imgIn, img_size * sizeof(float), hipMemcpyDeviceToHost);
        hipMemcpy(imgInPad, d_imgInPad, pad_img_size * sizeof(float), hipMemcpyDeviceToHost);
        hipMemcpy(kernel, d_kernel, kn * sizeof(float), hipMemcpyDeviceToHost);
        cv::Mat mImgOld(h, w, mIn.type());
        cv::Mat mImgPadOld(padh, padw, mIn.type());
        cv::Mat mKernelOld(nk, mk, CV_32F);
        //update the dimensions
        w = wP[level]; h = hP[level];
        mk = mP[level]; nk = nP[level];
        padw = w + mk - 1;
        padh = h + nk - 1;
        img_size = w * h * nc;
        pad_img_size = padw * padh * nc;
        kn = mk * nk;
        lambda = lambdas[level];
        //convert array to image
        cv::Mat mImgResize(h, w, mIn.type());
        cv::Mat mImgPadResize(padh, padw, mIn.type());
        cv::Mat mKernelResize(nk, mk, CV_32F);
        convertLayeredToMat(mImgOld, imgIn);
        convertLayeredToMat(mImgPadOld, imgInPad);
        convertLayeredToMat(mKernelOld, kernel);
        //Display image of previous iteration
        //convert Layered to mat
        //resize image
        cv::resize(mImgOld, mImgResize, mImgResize.size(), 0, 0, INTERPOLATION_METHOD);
        cv::resize(mImgPadOld, mImgPadResize, mImgPadResize.size(), 0, 0, INTERPOLATION_METHOD);
        cv::resize(mKernelOld, mKernelResize, mKernelResize.size(), 0, 0, INTERPOLATION_METHOD);
        //Copy image to array
        /*showImage("Image resized", mImgResize, 200, 200);*/
        /*showImage("Padded resized", mImgPadResize, 100, 200);*/
        /*cv::Mat mKernelOut(nk, mk, CV_8UC1);*/
        /*mKernelResize.convertTo(mKernelOut, CV_8UC1, 255.0 * (mk*nk), 0); */
        /*cv::namedWindow("Kernel", CV_WINDOW_NORMAL);*/
        /*cv::imshow("Kernel", mKernelOut);*/
        std::string img_name = "Image_" + std::to_string(level);
        std::string kernel_name = "Kernel_" + std::to_string(level);
        
        //copy image to GPU
        convertMatToLayered(imgIn, mImgResize);
        convertMatToLayered(imgInPad, mImgPadResize);
        convertMatToLayered(kernel, mKernelResize);
        saveMatrixMatlab(img_name.c_str(),
                        imgInPad,
                        padw,
                        padh,
                        nc);
        PyErr_Print();
        saveMatrixMatlab(kernel_name.c_str(),
                        kernel,
                        mk,
                        nk,
                        1);
        PyErr_Print();
        hipMemcpy(d_kernel, kernel, kn * sizeof(float), hipMemcpyHostToDevice);
        hipMemcpy(d_imgIn, imgIn, img_size * sizeof(float), hipMemcpyHostToDevice);
        hipMemcpy(d_imgInPad, imgInPad, pad_img_size * sizeof(float), hipMemcpyHostToDevice);
        hipMemcpy(d_imgInPadTemp, imgInPad, pad_img_size * sizeof(float), hipMemcpyHostToDevice);

        //select non zero kernel
        selectNonZeroGlobalMemCuda(d_kernel, mk, nk);
        /*hipDeviceSynchronize();*/

        //normalise kernel
        normaliseGlobalMemCuda(d_kernel, mk, nk);
        /*hipDeviceSynchronize();*/

        /*break;*/
    //}
    

    //Declaration of CUDNN descriptors
    hipdnnTensorDescriptor_t input_descriptor1, input_descriptor2, input_descriptor3;
    hipdnnFilterDescriptor_t kernel_descriptor1, kernel_descriptor2, kernel_descriptor3;
    hipdnnConvolutionDescriptor_t convolution_descriptor1, convolution_descriptor2, convolution_descriptor3;
    hipdnnTensorDescriptor_t output_descriptor1, output_descriptor2, output_descriptor3;
    hipdnnConvolutionFwdAlgo_t convolution_algorithm1, convolution_algorithm2, convolution_algorithm3;

    //workspace for cudnn
    void* d_workspace1{nullptr};
    void* d_workspace2{nullptr};
    void* d_workspace3{nullptr};

    //Defining Descriptors
    size_t workspace_bytes1 = createDescriptorsdc0(input_descriptor1,
                                               kernel_descriptor1,
                                               convolution_descriptor1,
                                               output_descriptor1,
                                               convolution_algorithm1,
                                               cudnn,
                                               padw,
                                               padh,
                                               mk,
                                               nk,
                                               nc);
    hipMalloc(&d_workspace1, workspace_bytes1);

    size_t workspace_bytes2 = createDescriptorsUp(input_descriptor2,
                                               kernel_descriptor2,
                                               convolution_descriptor2,
                                               output_descriptor2,
                                               convolution_algorithm2,
                                               cudnn,
                                               w,
                                               h,
                                               mk,
                                               nk,
                                               nc);
    hipMalloc(&d_workspace2, workspace_bytes2);

    size_t workspace_bytes3 = createDescriptorsdc1(input_descriptor3,
                                               kernel_descriptor3,
                                               convolution_descriptor3,
                                               output_descriptor3,
                                               convolution_algorithm3,
                                               cudnn,
                                               padw,
                                               padh,
                                               w,
                                               h,
                                               nc);
    hipMalloc(&d_workspace3, workspace_bytes3);
   
    // Blind Deconvolution algorithm starts here

    for(int iterations = 0; iterations < iter; ++iterations){
        //if(iterations % 100 == 0)
            std::cout << "Iteration num:  " << iterations << std::endl;

        // TODO: compute(mirror, rotate) kernel
        rotateKernel_180(d_kernel_temp, d_kernel, mk, nk); 
        /*hipDeviceSynchronize();*/
        //Our code
        /*computeDownConvolutionGlobalMemCuda(d_imgDownConv0, */
                                            /*d_imgInPadTemp, */
                                            /*d_kernel_temp, */
                                            /*padw, */
                                            /*padh, */
                                            /*nc, */
                                            /*mk, */
                                            /*nk);*/
        //CUDNN
        callConvolutiondc0(d_imgDownConv0,
                         d_imgInPadTemp,
                         d_kernel_temp,
                         padw, padh,
                         mk, nk,
                         w,h,nc,
                         input_descriptor1,
                         kernel_descriptor1,
                         convolution_descriptor1,
                         output_descriptor1,
                         convolution_algorithm1,
                         d_workspace1,
                         cudnn,
                         workspace_bytes1);
        /*hipMemcpy(imgUpConv, d_imgDownConv0, img_size * sizeof(float), hipMemcpyDeviceToHost);*/
    /*saveMatrixMatlab("cuda_conv1",*/
                    /*imgUpConv,*/
                    /*w,*/
                    /*h,*/
                    /*nc);*/
    /*PyErr_Print();*/
        /*hipDeviceSynchronize();*/
        /*if(iterations == 1)*/
            /*break;*/
        // DONE: cublas subtract k(+)*u - f. Move that to a separate function
        alpha = -1.0f;
        hipblasSetPointerMode(handle, HIPBLAS_POINTER_MODE_HOST);
        hipblasSaxpy(handle, img_size, &alpha, d_imgIn, 1, d_imgDownConv0, 1); CUDA_CHECK;
        /*hipDeviceSynchronize();*/
        
        //Our Call
        /*computeUpConvolutionGlobalMemCuda(d_imgUpConv, d_imgDownConv0, d_kernel, w, h, nc, mk, nk);*/

        //CUDNN
        callConvolutionUp(d_imgUpConv,
                         d_imgDownConv0,
                         d_kernel,
                         w, h,
                         mk, nk,
                         padw, padh, nc,
                         input_descriptor2,
                         kernel_descriptor2,
                         convolution_descriptor2,
                         output_descriptor2,
                         convolution_algorithm2,
                         d_workspace2,
                         cudnn,
                         workspace_bytes2);

        /*hipMemcpy(imgUpConv, d_imgUpConv, pad_img_size * sizeof(float), hipMemcpyDeviceToHost);*/
    /*saveMatrixMatlab("cuda_conv2",*/
                    /*imgUpConv,*/
                    /*padw,*/
                    /*padh,*/
                    /*nc);*/
    /*PyErr_Print();*/
        /*hipDeviceSynchronize();*/

        // compute gradient and divergence
        computeDiffOperatorsCuda(d_div, 
                                 d_dx_fw, d_dy_fw,
                                 d_dx_bw, d_dy_bw,
                                 d_dx_mixed, d_dy_mixed, 
                                 d_imgInPadTemp, padw, padh, nc, eps);
        
        /*hipMemcpy(imgUpConv, d_div, pad_img_size * sizeof(float), hipMemcpyDeviceToHost);*/
    /*saveMatrixMatlab("cuda_div",*/
                    /*imgUpConv,*/
                    /*padw,*/
                    /*padh,*/
                    /*nc);*/
    /*PyErr_Print();*/
        /*hipDeviceSynchronize();*/
        

        // TODO: subtract the divergence from upconvolution result (RAVIL)
        alpha = -1.0f * lambda;
        hipblasSetPointerMode(handle, HIPBLAS_POINTER_MODE_HOST);
        hipblasSaxpy(handle, pad_img_size, &alpha, d_div, 1, d_imgUpConv, 1); CUDA_CHECK;
        /*hipDeviceSynchronize();*/
        /*hipMemcpy(imgUpConv, d_imgUpConv, pad_img_size * sizeof(float), hipMemcpyDeviceToHost);*/
    /*saveMatrixMatlab("cuda_subU",*/
                    /*imgUpConv,*/
                    /*padw,*/
                    /*padh,*/
                    /*nc);*/
    /*PyErr_Print();*/
        // TODO: compute epsilon on GPU
        computeEpsilonGlobalMemCuda(d_epsU, handle, d_imgInPadTemp, d_imgUpConv, pad_img_size, 5e-3);
        /*hipDeviceSynchronize();*/
        

        // TODO: update output image u = u - eps*grad
        hipblasSetPointerMode(handle, HIPBLAS_POINTER_MODE_DEVICE);
        hipblasSaxpy(handle, pad_img_size, d_epsU, d_imgUpConv, 1, d_imgInPadTemp, 1);
        /*hipDeviceSynchronize();*/

        hipMemcpy(&epsU, d_epsU, sizeof(float), hipMemcpyDeviceToHost);
        /*hipMemcpy(imgUpConv, d_imgInPadTemp, pad_img_size * sizeof(float), hipMemcpyDeviceToHost);*/
    /*saveMatrixMatlab("cuda_updateU",*/
                    /*imgUpConv,*/
                    /*padw,*/
                    /*padh,*/
                    /*nc);*/
    /*PyErr_Print();*/

        //convoluton of k^y*y^{t+1}
        //Our implementation
        /*computeDownConvolutionGlobalMemCuda(d_imgDownConv1, */
                                            /*d_imgInPadTemp, */
                                            /*d_kernel_temp, */
                                            /*padw, */
                                            /*padh, */
                                            /*nc, */
                                            /*mk, nk);*/
        //CUDNN
        callConvolutiondc0(d_imgDownConv1,
                         d_imgInPadTemp,
                         d_kernel_temp,
                         padw, padh,
                         mk, nk,
                         w,h,nc,
                         input_descriptor1,
                         kernel_descriptor1,
                         convolution_descriptor1,
                         output_descriptor1,
                         convolution_algorithm1,
                         d_workspace1,
                         cudnn,
                         workspace_bytes1);
        hipMemcpy(imgUpConv, d_imgDownConv1, img_size * sizeof(float), hipMemcpyDeviceToHost);
    /*saveMatrixMatlab("cuda_conv3",*/
                    /*imgUpConv,*/
                    /*w,*/
                    /*h,*/
                    /*nc);*/
    /*PyErr_Print();*/
        /*hipDeviceSynchronize();*/

        //Substraction with f
        alpha = -1.0f;
        hipblasSetPointerMode(handle, HIPBLAS_POINTER_MODE_HOST);
        hipblasSaxpy(handle, img_size, &alpha, d_imgIn, 1, d_imgDownConv1, 1); CUDA_CHECK;
        /*hipDeviceSynchronize();*/

        // flip image
        // Ravi has checked that rotation is correct
        for(int c = 0; c < nc; ++c){
            rotateKernel_180(&d_imgPadRot[c*padw*padh], &d_imgInPadTemp[c*padw*padh], 
                    padw, padh);
            /*hipDeviceSynchronize(); */
            rotateKernel_180(&d_imgDownConv1Rot[c*w*h], &d_imgDownConv1[c*w*h], w, h);
            /*hipDeviceSynchronize();*/
        }

        // TODO: perform convolution: k = u * u_pad
        //Our implementation
        computeImageConvolution(d_kernel_temp, mk, nk ,
                                d_imgDownConv1Rot, d_imgInBuffer, 
                                w, h, 
                                d_imgPadRot, padw, padh, 
                                nc); 
        //CUDNN
        /*callConvolutiondc1(d_kernel_temp,*/
                         /*d_imgPadRot,*/
                         /*d_imgDownConv1Rot,*/
                         /*input_descriptor3,*/
                         /*kernel_descriptor3,*/
                         /*convolution_descriptor3,*/
                         /*output_descriptor3,*/
                         /*convolution_algorithm3,*/
                         /*d_workspace3,*/
                         /*cudnn,*/
                         /*workspace_bytes3);*/

        /*hipDeviceSynchronize();*/
        
        /*hipMemcpy(imgUpConv, d_kernel_temp, kn * sizeof(float), hipMemcpyDeviceToHost);*/
    /*saveMatrixMatlab("cuda_conv4",*/
                    /*imgUpConv,*/
                    /*mk,*/
                    /*nk,*/
                    /*1);*/
    /*PyErr_Print();*/
        computeEpsilonGlobalMemCuda(d_epsK, handle, d_kernel, d_kernel_temp, kn, 1e-3);
        /*hipDeviceSynchronize();*/
        /*hipMemcpy(kernel, d_kernel_temp, kn*sizeof(float), hipMemcpyDeviceToHost);*/

        /*std::cout << "Grad k" << std::endl;*/
        /*for(int i = 0;  i < nk; ++i){*/
            /*for(int j = 0; j < mk; ++j){*/
                /*std::cout << kernel[j + i*mk] << "   ";*/
            /*}*/
            /*std::cout << std::endl;*/
        /*}*/

        //update kernel
        hipblasSetPointerMode(handle, HIPBLAS_POINTER_MODE_DEVICE);
        hipblasSaxpy(handle, kn, d_epsK, d_kernel_temp, 1, d_kernel, 1);
        /*hipDeviceSynchronize();*/

        /*hipMemcpy(kernel, d_kernel, kn*sizeof(float), hipMemcpyDeviceToHost);*/

        /*std::cout << "After time update" << std::endl;*/
        /*for(int i = 0;  i < nk; ++i){*/
            /*for(int j = 0; j < mk; ++j){*/
                /*std::cout << kernel[j + i*mk] << "   ";*/
            /*}*/
            /*std::cout << std::endl;*/
        /*}*/

        //select non zero kernel
        selectNonZeroGlobalMemCuda(d_kernel, mk, nk);
        /*hipDeviceSynchronize();*/

        //normalise kernel
        normaliseGlobalMemCuda(d_kernel, mk, nk);
        /*hipDeviceSynchronize();*/

        //update lambda
        /*lambda = 0.99f * lambda;*/
        /*if(lambda < lambda_min){*/
            /*lambda = lambda_min;*/
        /*}*/

    
    }
    //Dec
    
    for(int iterations = 0; iterations < iter; ++iterations){
        //if(iterations % 100 == 0)
            std::cout << "Dec Iteration num:  " << iterations << std::endl;

        // TODO: compute(mirror, rotate) kernel
        rotateKernel_180(d_kernel_temp, d_kernel, mk, nk); 
        /*hipDeviceSynchronize();*/
        //Our code
        /*computeDownConvolutionGlobalMemCuda(d_imgDownConv0, */
                                            /*d_imgInPad, */
                                            /*d_kernel_temp, */
                                            /*padw, */
                                            /*padh, */
                                            /*nc, */
                                            /*mk, */
                                            /*nk);*/
        //CUDNN
        callConvolutiondc0(d_imgDownConv0,
                         d_imgInPad,
                         d_kernel_temp,
                         padw, padh,
                         mk, nk,
                         w,h,nc,
                         input_descriptor1,
                         kernel_descriptor1,
                         convolution_descriptor1,
                         output_descriptor1,
                         convolution_algorithm1,
                         d_workspace1,
                         cudnn,
                         workspace_bytes1);
        /*hipDeviceSynchronize();*/
        /*if(iterations == 1)*/
            /*break;*/
        // DONE: cublas subtract k(+)*u - f. Move that to a separate function
        alpha = -1.0f;
        hipblasSetPointerMode(handle, HIPBLAS_POINTER_MODE_HOST);
        hipblasSaxpy(handle, img_size, &alpha, d_imgIn, 1, d_imgDownConv0, 1); CUDA_CHECK;
        /*hipDeviceSynchronize();*/
        
        //Our Call
        /*computeUpConvolutionGlobalMemCuda(d_imgUpConv, d_imgDownConv0, d_kernel, w, h, nc, mk, nk);*/

        //CUDNN
        callConvolutionUp(d_imgUpConv,
                         d_imgDownConv0,
                         d_kernel,
                         w, h,
                         mk, nk,
                         padw, padh, nc,
                         input_descriptor2,
                         kernel_descriptor2,
                         convolution_descriptor2,
                         output_descriptor2,
                         convolution_algorithm2,
                         d_workspace2,
                         cudnn,
                         workspace_bytes2);

        /*hipDeviceSynchronize();*/

        // compute gradient and divergence
        computeDiffOperatorsCuda(d_div, 
                                 d_dx_fw, d_dy_fw,
                                 d_dx_bw, d_dy_bw,
                                 d_dx_mixed, d_dy_mixed, 
                                 d_imgInPad, padw, padh, nc, eps);
        /*hipDeviceSynchronize();*/
        

        // TODO: subtract the divergence from upconvolution result (RAVIL)
        alpha = -1.0f * lambda;

        hipblasSaxpy(handle, pad_img_size, &alpha, d_div, 1, d_imgUpConv, 1); CUDA_CHECK;
        // TODO: compute epsilon on GPU
        computeEpsilonGlobalMemCuda(d_epsU, handle, d_imgInPad, d_imgUpConv, pad_img_size, 5e-3);
        /*hipDeviceSynchronize();*/
        

        // TODO: update output image u = u - eps*grad
        hipblasSetPointerMode(handle, HIPBLAS_POINTER_MODE_DEVICE);
        hipblasSaxpy(handle, pad_img_size, d_epsU, d_imgUpConv, 1, d_imgInPad, 1);
        /*hipDeviceSynchronize();*/

    }
    destroyDescriptors(input_descriptor1,
                       kernel_descriptor1,
                       convolution_descriptor1,
                       output_descriptor1);
    hipFree(d_workspace1);
    destroyDescriptors(input_descriptor2,
                       kernel_descriptor2,
                       convolution_descriptor2,
                       output_descriptor2);
    hipFree(d_workspace2);
    destroyDescriptors(input_descriptor3,
                       kernel_descriptor3,
                       convolution_descriptor3,
                       output_descriptor3);
    hipFree(d_workspace3);
   }
	// show input image
    cv::Mat m_dx(padh, padw, mIn.type());
    cv::Mat m_dy(padh, padw, mIn.type());
    cv::Mat m_div(padh, padw, mIn.type());
    cv::Mat mPadImg(padh, padw, mIn.type());
    cv::Mat mImgDownConv0(h, w, mIn.type());
    cv::Mat mImgUpConv(padh, padw, mIn.type());
    cv::Mat mKernel(nk, mk, CV_32F);
    
    hipMemcpy(&epsU, d_epsU, sizeof(float), hipMemcpyDeviceToHost);
    hipMemcpy(&epsK, d_epsK, sizeof(float), hipMemcpyDeviceToHost);
    hipMemcpy(dx_fw, d_dx_fw, pad_img_size * sizeof(float), hipMemcpyDeviceToHost); CUDA_CHECK;
    hipMemcpy(imgInPad, d_imgInPad, pad_img_size* sizeof(float), hipMemcpyDeviceToHost); CUDA_CHECK;
    hipMemcpy(dy_fw, d_dy_fw, pad_img_size * sizeof(float), hipMemcpyDeviceToHost); CUDA_CHECK;

    hipMemcpy(dx_bw, d_dx_bw, pad_img_size * sizeof(float), hipMemcpyDeviceToHost); CUDA_CHECK;
    hipMemcpy(dy_bw, d_dy_bw, pad_img_size * sizeof(float), hipMemcpyDeviceToHost); CUDA_CHECK;

    hipMemcpy(dx_mixed, d_dx_mixed, pad_img_size * sizeof(float), hipMemcpyDeviceToHost);
    CUDA_CHECK;

hipMemcpy(dy_mixed, d_dy_mixed, pad_img_size * sizeof(float), hipMemcpyDeviceToHost);
    CUDA_CHECK;

    hipMemcpy(imgDownConv0, d_imgDownConv0, img_size * sizeof(float), hipMemcpyDeviceToHost);
    CUDA_CHECK;
    
    hipMemcpy(kernel, d_kernel, kn * sizeof(float), hipMemcpyDeviceToHost);
    CUDA_CHECK;

    hipMemcpy(div, d_div, pad_img_size * sizeof(float), hipMemcpyDeviceToHost); CUDA_CHECK;
    hipMemcpy(imgUpConv, d_imgUpConv, pad_img_size * sizeof(float), hipMemcpyDeviceToHost);
    CUDA_CHECK;

    //copy data from GPU to CPU
    /*float scale = 10.0;*/
    /*for (size_t i = 0; i < pad_img_size; ++i) {*/
        /*dx_fw[i] *= scale;*/
        /*dy_fw[i] *= scale;*/

        /*dx_bw[i] *= scale;*/
        /*dy_bw[i] *= scale;*/

        /*dx_mixed[i] *= scale;*/
        /*dy_mixed[i] *= scale;*/
        /*div[i] *= scale;*/
        /*std::cout << i << "  ---  " << dx_fw[i] << std::endl;*/
    /*}*/
    //DEBUG STARTS
    /*simpleTest(argc, argv);*/
    std::cout << "Value of epsilonU: " << epsU << std::endl;
    std::cout << "Value of epsilonK: " << epsK << std::endl;
    std::cout << "Value of lambda at the end: " << lambda << std::endl;

    /*for(int i = 0;  i < nk; ++i){*/
        /*for(int j = 0; j < mk; ++j){*/
            /*std::cout << std::setprecision(7) << kernel[j + i*mk] << "   ";*/
        /*}*/
        /*std::cout << std::endl;*/
    /*}*/
    /*std::cout << "YOLO" << std::endl; */
    /*for(int i = 0;  i < 5; ++i){*/
        /*for(int j = 0; j < 5; ++j){*/
            /*std::cout << std::setprecision(7) << imgDownConv0[j +  i*w] << "   ";*/
        /*}*/
        /*std::cout << std::endl;*/
    /*}*/

    //DEBUG ENDS
    

	// show output image: first convert to interleaved opencv format from the layered raw array
    convertLayeredToMat(m_dx, dx_mixed); 
    convertLayeredToMat(m_dy, dy_mixed); 
    convertLayeredToMat(m_div, div);
    convertLayeredToMat(mPadImg, imgInPad);
    convertLayeredToMat(mImgDownConv0, imgDownConv0);
    convertLayeredToMat(mImgUpConv, imgUpConv);
    convertLayeredToMat(mKernel, kernel);

    
    /*saveMatrixMatlab("interp_image",*/
                    /*imgIn,*/
                    /*w,*/
                    /*h,*/
                    /*nc);*/
    /*PyErr_Print();*/
    saveMatrixMatlab("output_image",
                    imgInPad,
                    padw,
                    padh,
                    nc);
    PyErr_Print();
    saveMatrixMatlab("output_kernel",
                    kernel,
                    mk,
                    nk,
                    1);
    PyErr_Print();

    size_t pos_orig_x = 100, pos_orig_y = 50, shift_y = 50; 
    showImage("Input", mIn, pos_orig_x, pos_orig_y);
    /*showImage("dx", m_dx, pos_orig_x + w, pos_orig_y);*/
    /*showImage("dy", m_dy, pos_orig_x, pos_orig_y + w + shift_y);*/
    showImage("divergence", m_div, pos_orig_x + w, pos_orig_y + w + shift_y);
    showImage("Output Image", mPadImg, 100, 140);
    /*showImage("Kernel", mKernel*10000, 150, 200);*/
    cv::Mat mKernelOut(nk, mk, CV_8UC1);
    mKernel.convertTo(mKernelOut, CV_8UC1, 255.0 * (mk*nk), 0); 
    /*cv::applyColorMap(mKernelOut, mKernelOut, cv::COLORMAP_JET);*/
    cv::namedWindow("Kernel", CV_WINDOW_NORMAL);
    cv::imshow("Kernel", mKernelOut);
    /*showImage("Down Conv 0", mImgDownConv0, 200, 240);*/
    /*showImage("Up Conv", mImgUpConv, 300, 340);*/

	//convertLayeredToMat(mOut, imgOut);
	//showImage("Output", mOut, 100+w+40, 100);

    // save results
    /*cv::imwrite("image_input.png",mIn*255.f); */
    /*cv::imwrite("image_result.png",m_div*255.f);*/
    /*cv::imwrite("image_kernel.png",mKernel*255.f);*/
    /*cv::imwrite("out.png",mPadImg);*/

    cv::FileStorage file("Kernel.txt", cv::FileStorage::WRITE);
    file << "Kernel" << mKernel;
    cv::FileStorage file1("Output.txt", cv::FileStorage::WRITE);
    file1 << "Output" << mPadImg;

    cv::waitKey(0);

    // Free allocated arrays
    delete [] imgIn;
    delete [] imgInPad; 
    delete [] imgOut;

    delete [] dx_fw;
    delete [] dy_fw;
    delete [] dx_bw;
    delete [] dy_bw;
    delete [] dx_mixed;
    delete [] dy_mixed;

    delete [] imgDownConv0;
    delete [] imgUpConv;

    delete [] div;
    delete [] kernel;

    hipFree(d_imgIn); CUDA_CHECK;
    hipFree(d_imgInPad); CUDA_CHECK;
    hipFree(d_imgInPadTemp); CUDA_CHECK;
    hipFree(d_imgPadRot); CUDA_CHECK;
    hipFree(d_imgInBuffer); CUDA_CHECK;
    hipFree(d_imgOut); CUDA_CHECK;
    hipFree(d_kernel); CUDA_CHECK;
    hipFree(d_kernel_temp); CUDA_CHECK;

    hipFree(d_dx_fw); CUDA_CHECK;
    hipFree(d_dy_fw); CUDA_CHECK;
    hipFree(d_dx_bw); CUDA_CHECK;
    hipFree(d_dy_bw); CUDA_CHECK;
    hipFree(d_dx_mixed); CUDA_CHECK;
    hipFree(d_dy_mixed); CUDA_CHECK;

    hipFree(d_imgDownConv0); CUDA_CHECK;
    hipFree(d_imgDownConv1); CUDA_CHECK;
    hipFree(d_imgDownConv1Rot); CUDA_CHECK;
    hipFree(d_imgUpConv); CUDA_CHECK;
    hipFree(d_epsU); CUDA_CHECK;
    hipFree(d_epsK); CUDA_CHECK;

    hipFree(d_div); CUDA_CHECK;

    hipblasDestroy(handle);
    hipdnnDestroy(cudnn);

    // close all opencv windows
    cv::destroyAllWindows();
    Py_Finalize();

    return 0;
} 




