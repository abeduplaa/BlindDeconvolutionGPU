// ########################################################################
// Practical Course: GPU Programming in Computer Vision
// Technical University of Munich, Computer Vision Group
// ########################################################################

#include <iostream>
#include <iomanip>
#include <string>
#include <opencv2/core/core.hpp>
#include <opencv2/highgui/highgui.hpp>
#include <opencv2/imgproc/imgproc.hpp>

#include "test.cuh"
#include "helper.cuh"
#include "downConvolution.cuh"
#include "testdownConv.cuh"
#include "divergence.cuh"
#include "adjustImage.cuh"
#include "pad.cuh"
#include "upConvolution.cuh"
#include "epsilon.cuh"
#include "selectNonZero.cuh"
#include "normalise.cuh"
#include "buildPyramid.cuh"
#include "cudnnDownConvolution.cuh"
#include "cudnnUpConvolution.cuh"

#include "hipblas.h"

#define INTERPOLATION_METHOD CV_INTER_CUBIC
//#define INTERPOLATION_METHOD CV_LINEAR

/*int main(int argc,char **argv)*/
/*{*/
    /*downConvTest();*/
/*}*/



int main(int argc,char **argv) {

	// TODO: ADD COMMAND LINE FUNCTIONS LATER

	// parse command line parameters
	const char *params = {
		"{image| |input image}"
		"{bw|false|load input image as grayscale/black-white}"
		"{mk|5|kernel width }"
		"{nk|5|kernel height}"
		"{cpu|false|compute on CPU}"
		"{eps|1e-3| epsilon }"
		"{lambda|0.0068|lambda }"
		"{iter|1| iter}"
		"{pad|0| pad type}"
	   // "{m|mem|0|memory: 0=global, 1=shared, 2=texture}"
	};
	cv::CommandLineParser cmd(argc, argv, params);

	// input image
	std::string inputImage = cmd.get<std::string>("image");
	// number of computation repetitions to get a better run time measurement
	// size_t repeats = (size_t)cmd.get<int>("repeats");
	// load the input image as grayscale
	 bool gray = cmd.get<bool>("bw");
	 int mk = cmd.get<int>("mk"); 
	 mk = (mk <= 0) ? 5 : mk;
	 int nk = cmd.get<int>("nk"); nk = (nk <= 0) ? 5 : nk;
	 bool is_cpu = cmd.get<bool>("cpu");
	 float lambda = cmd.get<float>("lambda"); 
	 std::cout << "lambda" << lambda << std::endl;
	 lambda = (lambda <= 0) ? 0.00035 : lambda; 
	 float lambda_min = 0.00035f;
	 float eps = cmd.get<float>("eps"); eps = ( eps <= 0 ) ? 1e-3 : eps;
	 int iter = cmd.get<int>("iter"); iter = ( iter <= 0 ) ? 1 : iter;
	 int padType = cmd.get<int>("pad"); padType = ( padType < 0 ) ? 0 : padType;

	 std::cout << "mode: " << (is_cpu ? "CPU" : "GPU") << std::endl;

	// TODO: LOAD IMAGE
	// read input frame
	cv::Mat mIn;
	// load the input image using opencv (load as grayscale if "gray==true", 
	// otherwise as is (may be color or grayscale))
	mIn = cv::imread(inputImage.c_str(), (gray ? CV_LOAD_IMAGE_GRAYSCALE : -1));

	// check
	if (mIn.empty()) {
		std::cerr << "ERROR: Could not retrieve frame " << inputImage << std::endl;
		return 1;
	}
	// convert to float representation (opencv loads image values as single bytes by default)
	mIn.convertTo(mIn, CV_32F);

	// get image dimensions
	int w = mIn.cols;         // width
	int h = mIn.rows;         // height
	int nc = mIn.channels();  // number of channels
	std::cout << "Original Image: " << w << " x " << h  << " x " << nc << std::endl;
	adjustImageSizeToOdd(mIn, w, h, nc);

	size_t img_size = w * h * nc;
	size_t padw = w + mk - 1;
	size_t padh = h + nk - 1;
	size_t pad_img_size = padw * padh * nc;
	std::cout << "Image after Cropping: " << w << " x " << h  << " x " << nc << std::endl;
	std::cout << "Pad Image size" << padw << " x " << padh << " x " << nc << std::endl;

	// init kernel
	size_t kn = mk*nk;
	float *kernel = new float[kn * sizeof(float)];
	//  initialize kernel to uniform.
	initialiseKernel(kernel, mk, nk);

	// initialize CUDA context
	// hipDeviceSynchronize();

	// Initialize Cublas
	hipblasHandle_t handle;
	hipblasCreate(&handle);

	//Initialize CUDNN
	hipdnnHandle_t cudnn;
	hipdnnCreate(&cudnn);

	//Pyramid parameter
	int pyramidSize =  0;
	const int smallestKernel = 3;
	const float kernelMultiplier = 1.1f;
	const float lambdaMultiplier = 1.9f;
	const float largestLambda = .11f;
	pyramidSize = pyramidScale(mk,nk, smallestKernel, kernelMultiplier, lambdaMultiplier, lambda, largestLambda);
	std::cout<< "Pyramid Size: " << pyramidSize << std::endl;
	// Build pyramid paramters:
	int *wP = new int[pyramidSize];
	int *hP = new int[pyramidSize];
	int *mP = new int[pyramidSize];
	int *nP = new int[pyramidSize];
	float *lambdas = new float[pyramidSize];

	buildPyramid1(wP, hP, mP, nP, lambdas, w, h, mk, nk, smallestKernel, kernelMultiplier, lambdaMultiplier, lambda, pyramidSize);


	// DEBUGGING:
	std::cout<< "pyramid facts: mP:  ";
	for(int i=0; i<pyramidSize; i++)
	{
		std::cout<< mP[i] << ", ";
	}
	std::cout<< "\n";
	std::cout<< "pyramid facts: nP:  " ;
	for(int i=0; i<pyramidSize; i++)
	{
		std::cout<< nP[i] << ", ";
	}
	std::cout<< "\n";
	std::cout<< "pyramid facts: wP:  " ;
	for(int i=0; i<pyramidSize; i++)
	{
		std::cout<< wP[i] << ", ";
	}
	std::cout<< "\n";
	std::cout<< "pyramid facts: hP:  ";
	for(int i=0; i<pyramidSize; i++)
	{
		std::cout<< hP[i] << ", ";
	}
	std::cout<< "\n";
	std::cout<< "pyramid facts: lambdas:  ";
	for(int i=0; i<pyramidSize; i++)
	{
		std::cout<< lambdas[i] << ", ";
	}
	std::cout<< "\n";


	// ### Set the output image format
	cv::Mat mOut(h, w, mIn.type());  // grayscale or color depending on input image, nc layers

	// ### Allocate arrays
	float *imgIn = new float[img_size];
	float *imgInPad = new float[pad_img_size]; //TODO: check size (RAVI)
	float *imgOut = new float[pad_img_size];
	float *dx_fw = new float[pad_img_size];
	float *dy_fw = new float[pad_img_size];
	float *dx_bw = new float[pad_img_size];
	float *dy_bw = new float[pad_img_size];
	float *dx_mixed = new float[pad_img_size];
	float *dy_mixed = new float[pad_img_size];
	float *imgDownConv0 = new float[img_size];
	float *imgUpConv = new float[pad_img_size];

	float *div = new float[pad_img_size];
	float epsU, epsK = 0.0f;
	float alpha = -1.0f;


	// TODO: ALLOCATE MEMORY ON GPU
	// allocate arrays on GPU
	float *d_imgIn = NULL;
	float *d_imgInPad = NULL;
	float *d_imgInBuffer = NULL;
	float *d_imgPadRot = NULL;
	float *d_imgOut = NULL;

	float *d_dx_fw = NULL;
	float *d_dy_fw = NULL;
	float *d_dx_bw = NULL;
	float *d_dy_bw = NULL;
	float *d_dx_mixed = NULL;
	float *d_dy_mixed = NULL;

	float *d_imgDownConv0 = NULL;
	float *d_imgDownConv1 = NULL;

	float *d_imgDownConv1Rot = NULL;

	float *d_imgUpConv = NULL;

	float *d_div = NULL;
	float *d_kernel = NULL;
	float *d_kernel_temp = NULL;

	float *d_epsU = NULL;
	float *d_epsK = NULL;

	hipMalloc(&d_imgIn, img_size * sizeof(float)); CUDA_CHECK;
	hipMalloc(&d_imgInBuffer, img_size * sizeof(float)); CUDA_CHECK;

	// TODO: be sure the size id right (RAVI)
	hipMalloc(&d_imgInPad, pad_img_size * sizeof(float)); CUDA_CHECK;
	hipMalloc(&d_imgPadRot, pad_img_size * sizeof(float)); CUDA_CHECK;
	hipMalloc(&d_imgOut , pad_img_size * sizeof(float)); CUDA_CHECK;

	hipMalloc(&d_kernel, kn * sizeof(float)); CUDA_CHECK;
	hipMalloc(&d_kernel_temp, kn  * sizeof(float)); CUDA_CHECK;

	hipMalloc(&d_dx_fw, pad_img_size * sizeof(float)); CUDA_CHECK;
	hipMalloc(&d_dy_fw, pad_img_size * sizeof(float)); CUDA_CHECK;

	hipMalloc(&d_dx_bw, pad_img_size * sizeof(float)); CUDA_CHECK;
	hipMalloc(&d_dy_bw, pad_img_size * sizeof(float)); CUDA_CHECK;

	hipMalloc(&d_dx_mixed, pad_img_size * sizeof(float)); CUDA_CHECK;
	hipMalloc(&d_dy_mixed, pad_img_size * sizeof(float)); CUDA_CHECK;

	hipMalloc(&d_div , pad_img_size * sizeof(float)); CUDA_CHECK;

	hipMalloc(&d_imgDownConv0, img_size * sizeof(float)); CUDA_CHECK;
	hipMalloc(&d_imgDownConv1, img_size * sizeof(float)); CUDA_CHECK;

	hipMalloc(&d_imgDownConv1Rot, img_size * sizeof(float)); CUDA_CHECK;

	hipMalloc(&d_imgUpConv, pad_img_size * sizeof(float)); CUDA_CHECK;
	hipMalloc(&d_epsU, sizeof(float)); CUDA_CHECK;
	hipMalloc(&d_epsK, sizeof(float)); CUDA_CHECK;

	mIn /= 255.0f;
	convertMatToLayered(imgIn, mIn);
	hipMemcpy(d_imgIn, imgIn, img_size * sizeof(float), hipMemcpyHostToDevice); CUDA_CHECK;
	hipMemcpy(d_kernel, kernel, kn * sizeof(float), hipMemcpyHostToDevice); CUDA_CHECK;

	padImgGlobalMemCuda(d_imgInPad, d_imgIn, w, h, nc, mk, nk,padType);
	//Start loop  for pyramid

	int START_LEVEL = 0;
	for(int level = START_LEVEL; level >= 0; --level){
		//copy image and padded image back to CPU
		std::cout << "Level Number:   " << level << std::endl;
		hipMemcpy(imgIn, d_imgIn, img_size * sizeof(float), hipMemcpyDeviceToHost);
		hipMemcpy(imgInPad, d_imgInPad, pad_img_size * sizeof(float), hipMemcpyDeviceToHost);
		hipMemcpy(kernel, d_kernel, kn * sizeof(float), hipMemcpyDeviceToHost);
		cv::Mat mImgOld(h, w, mIn.type());
		cv::Mat mImgPadOld(padh, padw, mIn.type());
		cv::Mat mKernelOld(nk, mk, CV_32F);
		//update the dimensions
		w = wP[level]; h = hP[level];
		mk = mP[level]; nk = nP[level];
		padw = w + mk - 1;
		padh = h + nk - 1;
		img_size = w * h * nc;
		pad_img_size = padw * padh * nc;
		kn = mk * nk;
		//convert array to image
		cv::Mat mImgResize(h, w, mIn.type());
		cv::Mat mImgPadResize(padh, padw, mIn.type());
		cv::Mat mKernelResize(nk, mk, CV_32F);
		convertLayeredToMat(mImgOld, imgIn);
		convertLayeredToMat(mImgPadOld, imgInPad);
		convertLayeredToMat(mKernelOld, kernel);
		//Display image of previous iteration
		//convert Layered to mat
		//resize image
		cv::resize(mImgOld, mImgResize, mImgResize.size(), 0, 0, INTERPOLATION_METHOD);
		cv::resize(mImgPadOld, mImgPadResize, mImgPadResize.size(), 0, 0, INTERPOLATION_METHOD);
		cv::resize(mKernelOld, mKernelResize, mKernelResize.size(), 0, 0, INTERPOLATION_METHOD);
		//Copy image to array
		showImage("Image resized", mImgResize, 200, 200);
		showImage("Padded resized", mImgPadResize, 100, 200);
		showImage("Kernel resized", mKernelResize * 100, 200, 200);
		cv::waitKey(0);
		
		//copy image to GPU
		convertMatToLayered(imgIn, mImgResize);
		convertMatToLayered(imgInPad, mImgPadResize);
		convertMatToLayered(kernel, mKernelResize);
		hipMemcpy(d_kernel, kernel, kn * sizeof(float), hipMemcpyHostToDevice);
		hipMemcpy(d_imgIn, imgIn, img_size * sizeof(float), hipMemcpyHostToDevice);
		hipMemcpy(d_imgInPad, imgInPad, pad_img_size * sizeof(float), hipMemcpyHostToDevice);
	//}


	//Declaration of CUDNN descriptors
	hipdnnTensorDescriptor_t input_descriptor1, input_descriptor2, input_descriptor3;
	hipdnnFilterDescriptor_t kernel_descriptor1, kernel_descriptor2, kernel_descriptor3;
	hipdnnConvolutionDescriptor_t convolution_descriptor1, convolution_descriptor2, convolution_descriptor3;
	hipdnnTensorDescriptor_t output_descriptor1, output_descriptor2, output_descriptor3;
	hipdnnConvolutionFwdAlgo_t convolution_algorithm1, convolution_algorithm2, convolution_algorithm3;

	//workspace for cudnn
	void* d_workspace1{nullptr};
	void* d_workspace2{nullptr};
	void* d_workspace3{nullptr};

	//Defining Descriptors
	size_t workspace_bytes1 = createDescriptorsdc0(input_descriptor1,
											   kernel_descriptor1,
											   convolution_descriptor1,
											   output_descriptor1,
											   convolution_algorithm1,
											   cudnn,
											   padw,
											   padh,
											   mk,
											   nk,
											   nc);
	hipMalloc(&d_workspace1, workspace_bytes1);

	size_t workspace_bytes2 = createDescriptorsUp(input_descriptor2,
											   kernel_descriptor2,
											   convolution_descriptor2,
											   output_descriptor2,
											   convolution_algorithm2,
											   cudnn,
											   w,
											   h,
											   mk,
											   nk,
											   nc);
	hipMalloc(&d_workspace2, workspace_bytes2);

	size_t workspace_bytes3 = createDescriptorsdc1(input_descriptor3,
											   kernel_descriptor3,
											   convolution_descriptor3,
											   output_descriptor3,
											   convolution_algorithm3,
											   cudnn,
											   padw,
											   padh,
											   w,
											   h,
											   nc);
	hipMalloc(&d_workspace3, workspace_bytes3);

	for(int iterations = 0; iterations < iter; ++iterations){
		std::cout << "Iteration num:  " << iterations << std::endl;

		// TODO: compute(mirror, rotate) kernel
		rotateKernel_180(d_kernel_temp, d_kernel, mk, nk); 
		hipDeviceSynchronize();
		//Our code
		/*computeDownConvolutionGlobalMemCuda(d_imgDownConv0, */
											/*d_imgInPad, */
											/*d_kernel_temp, */
											/*padw, */
											/*padh, */
											/*nc, */
											/*mk, */
											/*nk);*/
		//CUDNN
		callConvolutiondc0(d_imgDownConv0,
						 d_imgInPad,
						 d_kernel_temp,
						 padw, padh,
						 mk, nk,
						 w,h,nc,
						 input_descriptor1,
						 kernel_descriptor1,
						 convolution_descriptor1,
						 output_descriptor1,
						 convolution_algorithm1,
						 d_workspace1,
						 cudnn,
						 workspace_bytes1);
		hipDeviceSynchronize();
		/*if(iterations == 1)*/
			/*break;*/
		// DONE: cublas subtract k(+)*u - f. Move that to a separate function
		alpha = -1.0f;
		hipblasSetPointerMode(handle, HIPBLAS_POINTER_MODE_HOST);
		hipblasSaxpy(handle, img_size, &alpha, d_imgIn, 1, d_imgDownConv0, 1); CUDA_CHECK;
		hipDeviceSynchronize();
		
		//Our Call
		/*computeUpConvolutionGlobalMemCuda(d_imgUpConv, d_imgDownConv0, d_kernel, w, h, nc, mk, nk);*/

		//CUDNN
		callConvolutionUp(d_imgUpConv,
						 d_imgDownConv0,
						 d_kernel,
						 w, h,
						 mk, nk,
						 padw, padh, nc,
						 input_descriptor2,
						 kernel_descriptor2,
						 convolution_descriptor2,
						 output_descriptor2,
						 convolution_algorithm2,
						 d_workspace2,
						 cudnn,
						 workspace_bytes2);

		hipDeviceSynchronize();

		// compute gradient and divergence
		computeDiffOperatorsCuda(d_div, 
								 d_dx_fw, d_dy_fw,
								 d_dx_bw, d_dy_bw,
								 d_dx_mixed, d_dy_mixed, 
								 d_imgInPad, padw, padh, nc, 1.0f, eps);
		hipDeviceSynchronize();
		

		// TODO: subtract the divergence from upconvolution result (RAVIL)
		alpha = -1.0f * lambda;

		hipblasSaxpy(handle, pad_img_size, &alpha, d_div, 1, d_imgUpConv, 1); CUDA_CHECK;
		// TODO: compute epsilon on GPU
		computeEpsilonGlobalMemCuda(d_epsU, handle, d_imgInPad, d_imgUpConv, pad_img_size, 5e-3);
		hipDeviceSynchronize();
		

		// TODO: update output image u = u - eps*grad
		hipblasSetPointerMode(handle, HIPBLAS_POINTER_MODE_DEVICE);
		hipblasSaxpy(handle, pad_img_size, d_epsU, d_imgUpConv, 1, d_imgInPad, 1);
		hipDeviceSynchronize();


		//convoluton of k^y*y^{t+1}
		//Our implementation
		/*computeDownConvolutionGlobalMemCuda(d_imgDownConv1, */
											/*d_imgInPad, */
											/*d_kernel_temp, */
											/*padw, */
											/*padh, */
											/*nc, */
											/*mk, nk);*/
		//CUDNN
		callConvolutiondc0(d_imgDownConv1,
						 d_imgInPad,
						 d_kernel_temp,
						 padw, padh,
						 mk, nk,
						 w,h,nc,
						 input_descriptor1,
						 kernel_descriptor1,
						 convolution_descriptor1,
						 output_descriptor1,
						 convolution_algorithm1,
						 d_workspace1,
						 cudnn,
						 workspace_bytes1);
		hipDeviceSynchronize();

		//Substraction with f
		alpha = -1.0f;
		hipblasSetPointerMode(handle, HIPBLAS_POINTER_MODE_HOST);
		hipblasSaxpy(handle, img_size, &alpha, d_imgIn, 1, d_imgDownConv1, 1); CUDA_CHECK;
		hipDeviceSynchronize();

		// flip image
		// Ravi has checked that rotation is correct
		for(int c = 0; c < nc; ++c){
			rotateKernel_180(&d_imgPadRot[c*padw*padh], &d_imgInPad[c*padw*padh], 
					padw, padh);
		   hipDeviceSynchronize(); 
			rotateKernel_180(&d_imgDownConv1Rot[c*w*h], &d_imgDownConv1[c*w*h], w, h);
			hipDeviceSynchronize();
		}

		// TODO: perform convolution: k = u * u_pad
		//Our implementation
		/*computeImageConvolution(d_kernel_temp, mk, nk ,*/
								/*d_imgDownConv1Rot, d_imgInBuffer, */
								/*w, h, */
								/*d_imgPadRot, padw, padh, */
								/*nc); */
		//CUDNN
		callConvolutiondc1(d_kernel_temp,
						 d_imgPadRot,
						 d_imgDownConv1Rot,
						 input_descriptor3,
						 kernel_descriptor3,
						 convolution_descriptor3,
						 output_descriptor3,
						 convolution_algorithm3,
						 d_workspace3,
						 cudnn,
						 workspace_bytes3);

		hipDeviceSynchronize();
		
		computeEpsilonGlobalMemCuda(d_epsK, handle, d_kernel, d_kernel_temp, kn, 1e-3);
		hipDeviceSynchronize();
		/*hipMemcpy(kernel, d_kernel_temp, kn*sizeof(float), hipMemcpyDeviceToHost);*/

		/*std::cout << "Grad k" << std::endl;*/
		/*for(int i = 0;  i < nk; ++i){*/
			/*for(int j = 0; j < mk; ++j){*/
				/*std::cout << kernel[j + i*mk] << "   ";*/
			/*}*/
			/*std::cout << std::endl;*/
		/*}*/

		//update kernel
		hipblasSetPointerMode(handle, HIPBLAS_POINTER_MODE_DEVICE);
		hipblasSaxpy(handle, kn, d_epsK, d_kernel_temp, 1, d_kernel, 1);
		hipDeviceSynchronize();

		/*hipMemcpy(kernel, d_kernel, kn*sizeof(float), hipMemcpyDeviceToHost);*/

		/*std::cout << "After time update" << std::endl;*/
		/*for(int i = 0;  i < nk; ++i){*/
			/*for(int j = 0; j < mk; ++j){*/
				/*std::cout << kernel[j + i*mk] << "   ";*/
			/*}*/
			/*std::cout << std::endl;*/
		/*}*/

		//select non zero kernel
		selectNonZeroGlobalMemCuda(d_kernel, mk, nk);
		hipDeviceSynchronize();

		//normalise kernel
		normaliseGlobalMemCuda(d_kernel, mk, nk);
		hipDeviceSynchronize();

		//update lambda
		lambda = 0.99f * lambda;
		if(lambda < lambda_min){
			lambda = lambda_min;
		}


	}

	destroyDescriptors(input_descriptor1,
					   kernel_descriptor1,
					   convolution_descriptor1,
					   output_descriptor1);
	hipFree(d_workspace1);
	destroyDescriptors(input_descriptor2,
					   kernel_descriptor2,
					   convolution_descriptor2,
					   output_descriptor2);
	hipFree(d_workspace2);
	destroyDescriptors(input_descriptor3,
					   kernel_descriptor3,
					   convolution_descriptor3,
					   output_descriptor3);
	hipFree(d_workspace3);
	}
	// convert range of each channel to [0,1]
	// init raw input image array (and convert to layered)

	// TODO:  IMPLEMENT THESE FUNCTIONS

	// 1. pre-process: pad image


	// 2. perform blind de-convolution



	// padImage(imgIn);
	// computeDeconvolution(imgOut, imgIn, kernel, w, h, nc);


	//hipMemcpy(imgOut,d_imgOut,nbytes,hipMemcpyDeviceToHost);

	// show input image

	cv::Mat m_dx(padh, padw, mIn.type());
	cv::Mat m_dy(padh, padw, mIn.type());
	cv::Mat m_div(padh, padw, mIn.type());
	cv::Mat mPadImg(padh, padw, mIn.type());
	cv::Mat mImgDownConv0(h, w, mIn.type());
	cv::Mat mImgUpConv(padh, padw, mIn.type());
	cv::Mat mKernel(nk, mk, CV_32F);

	hipMemcpy(&epsU, d_epsU, sizeof(float), hipMemcpyDeviceToHost);
	hipMemcpy(&epsK, d_epsK, sizeof(float), hipMemcpyDeviceToHost);
	hipMemcpy(dx_fw, d_dx_fw, pad_img_size * sizeof(float), hipMemcpyDeviceToHost); CUDA_CHECK;
	hipMemcpy(imgInPad, d_imgInPad, pad_img_size* sizeof(float), hipMemcpyDeviceToHost); CUDA_CHECK;
	hipMemcpy(dy_fw, d_dy_fw, pad_img_size * sizeof(float), hipMemcpyDeviceToHost); CUDA_CHECK;

	hipMemcpy(dx_bw, d_dx_bw, pad_img_size * sizeof(float), hipMemcpyDeviceToHost); CUDA_CHECK;
	hipMemcpy(dy_bw, d_dy_bw, pad_img_size * sizeof(float), hipMemcpyDeviceToHost); CUDA_CHECK;

	hipMemcpy(dx_mixed, d_dx_mixed, pad_img_size * sizeof(float), hipMemcpyDeviceToHost);
	CUDA_CHECK;

	hipMemcpy(dy_mixed, d_dy_mixed, pad_img_size * sizeof(float), hipMemcpyDeviceToHost);
	CUDA_CHECK;

	hipMemcpy(imgDownConv0, d_imgDownConv0, img_size * sizeof(float), hipMemcpyDeviceToHost);
	CUDA_CHECK;

	hipMemcpy(kernel, d_kernel, kn * sizeof(float), hipMemcpyDeviceToHost);
	CUDA_CHECK;

	hipMemcpy(div, d_div, pad_img_size * sizeof(float), hipMemcpyDeviceToHost); CUDA_CHECK;
	hipMemcpy(imgUpConv, d_imgUpConv, pad_img_size * sizeof(float), hipMemcpyDeviceToHost);
	CUDA_CHECK;

	//copy data from GPU to CPU
	/*float scale = 10.0;*/
	/*for (size_t i = 0; i < pad_img_size; ++i) {*/
		/*dx_fw[i] *= scale;*/
		/*dy_fw[i] *= scale;*/

		/*dx_bw[i] *= scale;*/
		/*dy_bw[i] *= scale;*/

		/*dx_mixed[i] *= scale;*/
		/*dy_mixed[i] *= scale;*/
		/*div[i] *= scale;*/
		/*std::cout << i << "  ---  " << dx_fw[i] << std::endl;*/
	/*}*/
	//DEBUG STARTS
	/*simpleTest(argc, argv);*/
	std::cout << "Value of epsilonU: " << epsU << std::endl;
	std::cout << "Value of epsilonK: " << epsK << std::endl;
	std::cout << "Value of lambda at the end: " << lambda << std::endl;
	/*
	for(int i = 0;  i < nk; ++i){
		for(int j = 0; j < mk; ++j){
			std::cout << std::setprecision(7) << kernel[j + i*mk] << "   ";
		}
		std::cout << std::endl;
	}*/
	/*std::cout << "YOLO" << std::endl; */
	/*for(int i = 0;  i < 5; ++i){*/
		/*for(int j = 0; j < 5; ++j){*/
			/*std::cout << std::setprecision(7) << imgDownConv0[j +  i*w] << "   ";*/
		/*}*/
		/*std::cout << std::endl;*/
	/*}*/

	//DEBUG ENDS


	// show output image: first convert to interleaved opencv format from the layered raw array
	convertLayeredToMat(m_dx, dx_mixed); 
	convertLayeredToMat(m_dy, dy_mixed); 
	convertLayeredToMat(m_div, div);
	convertLayeredToMat(mPadImg, imgInPad);
	convertLayeredToMat(mImgDownConv0, imgDownConv0);
	convertLayeredToMat(mImgUpConv, imgUpConv);
	convertLayeredToMat(mKernel, kernel);

	size_t pos_orig_x = 100, pos_orig_y = 50, shift_y = 50; 
	showImage("Input", mIn, pos_orig_x, pos_orig_y);
	/*showImage("dx", m_dx, pos_orig_x + w, pos_orig_y);*/
	/*showImage("dy", m_dy, pos_orig_x, pos_orig_y + w + shift_y);*/
	showImage("divergence", m_div, pos_orig_x + w, pos_orig_y + w + shift_y);
	showImage("Output Image", mPadImg, 100, 140);
	showImage("Kernel", mKernel*10000, 150, 200);
	/*showImage("Down Conv 0", mImgDownConv0, 200, 240);*/
	/*showImage("Up Conv", mImgUpConv, 300, 340);*/

	//convertLayeredToMat(mOut, imgOut);
	//showImage("Output", mOut, 100+w+40, 100);

	// save results
	/*cv::imwrite("image_input.png",mIn*255.f); */
	/*cv::imwrite("image_result.png",m_div*255.f);*/
	/*cv::imwrite("image_kernel.png",mKernel*255.f);*/

	cv::waitKey(0);

	// Free allocated arrays
	delete [] imgIn;
	delete [] imgInPad; 
	delete [] imgOut;

	delete [] dx_fw;
	delete [] dy_fw;
	delete [] dx_bw;
	delete [] dy_bw;
	delete [] dx_mixed;
	delete [] dy_mixed;

	delete [] imgDownConv0;
	delete [] imgUpConv;

	delete [] div;
	delete [] kernel;

	hipFree(d_imgIn); CUDA_CHECK;
	hipFree(d_imgInPad); CUDA_CHECK;
	hipFree(d_imgPadRot); CUDA_CHECK;
	hipFree(d_imgInBuffer); CUDA_CHECK;
	hipFree(d_imgOut); CUDA_CHECK;
	hipFree(d_kernel); CUDA_CHECK;
	hipFree(d_kernel_temp); CUDA_CHECK;

	hipFree(d_dx_fw); CUDA_CHECK;
	hipFree(d_dy_fw); CUDA_CHECK;
	hipFree(d_dx_bw); CUDA_CHECK;
	hipFree(d_dy_bw); CUDA_CHECK;
	hipFree(d_dx_mixed); CUDA_CHECK;
	hipFree(d_dy_mixed); CUDA_CHECK;

	hipFree(d_imgDownConv0); CUDA_CHECK;
	hipFree(d_imgDownConv1); CUDA_CHECK;
	hipFree(d_imgDownConv1Rot); CUDA_CHECK;
	hipFree(d_imgUpConv); CUDA_CHECK;
	hipFree(d_epsU); CUDA_CHECK;
	hipFree(d_epsK); CUDA_CHECK;

	hipFree(d_div); CUDA_CHECK;

	hipblasDestroy(handle);
	hipdnnDestroy(cudnn);

	// close all opencv windows
	cv::destroyAllWindows();

	return 0;
} 
