#include "hip/hip_runtime.h"
#include "divergence.cuh"
#include "helper.cuh"

#include <iostream>
#include <stdio.h>
#include <hip/hip_runtime.h>
#include "helper.cuh"
#include <stdio.h>


__global__
void computeGradientsKernel(float *dx_fw, float *dy_fw,
                            float *dx_bw, float  *dy_bw,
                            float *dx_mixed, float *dy_mixed, 
                            const float *imgIn, const int w, const int h, const int nc) {

    for (int channel = 0; channel < nc; ++channel) {

        int offset = w * h * channel;
        int idx = threadIdx.x + blockIdx.x * blockDim.x;
        int idy = threadIdx.y + blockIdx.y * blockDim.y;

        // TODO: flip x and y loops
        for (int x = idx; x < w; x += blockDim.x * gridDim.x) {
            for (int y = idy; y < h; y += blockDim.y * gridDim.y) {
                float center = imgIn[getIndex(x, y, w) + offset];

                float right = (x < w) ? imgIn[getIndex(x + 1, y, w) + offset] : center;  
                float left = (x > 0) ? imgIn[getIndex(x - 1, y, w) + offset] : center;  

                float top = (y < h) 
                          ? imgIn[getIndex(x, y + 1, w) + offset] : center;  
                float down = (y > 0) 
                           ? imgIn[getIndex(x, y - 1, w) + offset] : center;  

                float top_left = ((x > 0) && (y < h - 1)) 
                                   ? imgIn[getIndex(x - 1, y + 1, w) + offset] : left;

                float down_right = ((x < w - 1) && (y > 0)) 
                                    ? imgIn[getIndex(x + 1, y - 1, w) + offset] : down;
                /*float top_right = ((x < w) && (y < h)) */
                               /*? imgIn[getIndex(x + 1, y + 1, w) + offset] : top;*/

                /*float down_left = ((x > 0) && (y > 0)) */
                                 /*? imgIn[getIndex(x - 1, y - 1, w) + offset] : left;*/

                dx_fw[getIndex(x, y, w) + offset] = right - center;
                dy_fw[getIndex(x, y, w) + offset] = top - center;

                dx_bw[getIndex(x, y, w) + offset] = center - left;
                dy_bw[getIndex(x, y, w) + offset] = center - down;

                dx_mixed[getIndex(x, y, w) + offset] = down_right - down;
                dy_mixed[getIndex(x, y, w) + offset] = top_left - left;
                /*dx_mixed[getIndex(x, y, w) + offset] = top_right - top;*/
                /*dy_mixed[getIndex(x, y, w) + offset] = down_left - left;*/
            }
        }
    }
}


__device__
float computeDuffusivityDevice(const float a, const float  b, const float eps) {
    float length = sqrt(a * a + b * b);
    return length > eps ? length : eps; 
}


__global__
void computeDivergenceKernel(float *div, 
                             const float *dx_fw, const float *dy_fw,
                             const float *dx_bw, const float *dy_bw,
                             const float *dx_mixed, const float *dy_mixed, 
                             const float *imgIn, const int w, const int h, const int nc,
                             const float eps) {

    if(threadIdx.x == 0 && threadIdx.y == 0 && blockIdx.x == 0 && blockIdx.y == 0)
        printf("EPS on gpu: %f\n", eps);

    for (int channel = 0; channel < nc; ++channel) {

        int offset = w * h * channel;
        int idx = threadIdx.x + blockIdx.x * blockDim.x;
        int idy = threadIdx.y + blockIdx.y * blockDim.y;

        for (int x = idx; x < w; x += blockDim.x * gridDim.x) {
            for (int y = idy; y < h; y += blockDim.y * gridDim.y) {
                int index = getIndex(x, y, w) + offset; 

                float norm_1 = computeDuffusivityDevice(dx_fw[index], dy_fw[index], eps); 
                float sum = dx_fw[index] + dy_fw[index];
                div[index] = sum / norm_1;

                /*div[index] = (dx_fw[index] + dy_fw[index]);*/

                /*div[index] = ((dx_fw[index] + dy_fw[index]) */
                                        /*/ computeDuffusivity(dx_fw[index], dy_fw[index], eps)) */
                           /*- dx_bw[index] / computeDuffusivity(dx_bw[index], dy_mixed[index], eps)*/
                           /*- dy_bw[index] / computeDuffusivity(dx_mixed[index], dy_bw[index], eps); */
            }
        }
    }
}


void computeDiffOperatorsCuda(float *d_div, 
                              float *d_dx_fw, float *d_dy_fw,
                              float *d_dx_bw, float *d_dy_bw,
                              float *d_dx_mixed, float *d_dy_mixed, 
                              const float *d_imgIn, const int w, const int h, const int nc,
                              const float eps) {
    dim3 block(32, 2, 1);
    dim3 grid = computeGrid2D(block, w, h);
    
    computeGradientsKernel<<<grid, block>>>(d_dx_fw, d_dy_fw,
                                            d_dx_bw, d_dy_bw,
                                            d_dx_mixed, d_dy_mixed, 
                                            d_imgIn, w, h, nc);

    hipDeviceSynchronize();
    computeDivergenceKernel<<<grid, block>>>(d_div, 
                                             d_dx_fw, d_dy_fw,
                                             d_dx_bw, d_dy_bw,
                                             d_dx_mixed, d_dy_mixed, 
                                             d_imgIn, w, h, nc,
                                             eps);
    CUDA_CHECK;
}

