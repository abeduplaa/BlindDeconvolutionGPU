#include "hip/hip_runtime.h"
#include "pad.cuh"

#include <iostream>
#include <hip/hip_runtime.h>
#include "helper.cuh"

__global__
void padImgGlobalMemKernel(float* imgOut, const float* imgIn,
        int w, int h, int nc, int padX, int padY){
    int idx = threadIdx.x + blockIdx.x*blockDim.x;
    int idy = threadIdx.y + blockIdx.y*blockDim.y;
    int id, tempX, tempY;
    int outSizeX = w + padX + padX;
    int outSizeY = h + padY + padY;

    for(int c = 0; c < nc; ++c){
        id = idx + (idy*outSizeX) + (c*outSizeX*outSizeY);
        if(idx < outSizeX && idy < outSizeY){
            tempX = idx - padX;
            tempY = idy - padY;
            if(tempX >= 0 && tempX < w && tempY >= 0 && tempY < h){
                imgOut[id] = imgIn[tempX + (tempY*w) +(c*w*h)];
            }
            else{
                imgOut[id] = 0.0f;
            }
        }
    }
}

void padImgCPU(float* imgOut, const float* imgIn,
               size_t w, size_t h, size_t nc,
               size_t m, size_t n){
    size_t padX = floor(m/2.0);
    size_t padY = floor(n/2.0);
    size_t outSizeX = w+padX+padX;
    size_t outSizeY = h+padY+padY;
    size_t totalSize = outSizeX * outSizeY*nc;
    for(int i=0; i<totalSize; ++i)
        imgOut[i] = 0.f;
    for(int c=0; c<nc; ++c){
        for(int j=0; j<h; ++j){
            for(int i=0; i<w; ++i){
                imgOut[padX + i + (j+padY)*outSizeX + c*outSizeX*outSizeY] = 
                    imgIn[i + j*w + c*w*h];
            }
        }
    }
}

void padImgGlobalMemCuda(float *imgOut, const float *imgIn,
                         int w, int h, int nc, int m, int n){
    if(!imgIn){
        std::cout<< " input not allocated"<<std::endl;
        return;
    }
    if(!imgOut){
        std::cout << "output not allocated" << std::endl;         
        return;   
    }
    if (!imgOut || !imgIn)
    {
        std::cerr << "arrays not allocated!" << std::endl;
        return;
                            
    }

    // allocate block and grid size
    dim3 block(32, 8, 1);
    int padX = (int) floor(m/2.0f);
    int padY = (int) floor(n/2.0f);
    dim3 grid = computeGrid2D(block, w + m - 1, h + n - 1);

    //calling cuda kernel
    padImgGlobalMemKernel <<<grid,block>>> (imgOut, imgIn, w, h, nc, padX, padY);
}
